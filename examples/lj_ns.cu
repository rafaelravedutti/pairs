#include "hip/hip_runtime.h"
#define PAIRS_TARGET_CUDA
#include <math.h>
#include <stdbool.h>
#include <stdio.h>
#include <stdlib.h>
//---
#include "runtime/pairs.hpp"
#include "runtime/read_from_file.hpp"
#include "runtime/vtk.hpp"

using namespace pairs;

__constant__ int d_dim_cells[3];

__global__ void pack_ghost_particles0_0_1_2_kernel0(int range_start, double grid0_d0_max, double grid0_d0_min, double grid0_d1_max, double grid0_d1_min, double grid0_d2_max, double grid0_d2_min, int *send_offsets, int *nsend, double *send_buffer, int *send_map, int *send_mult, double *mass, double *position, double *velocity, double e138, double e147, double e156) {
    const int i8 = blockIdx.x * blockDim.x + threadIdx.x + range_start;
    if((i8 < (send_offsets[0] + (nsend[0] + nsend[1])))) {
        const int e132 = i8 * 7;
        const int a79 = send_map[i8];
        const double p8 = mass[a79];
        send_buffer[e132] = p8;
        const int e141 = i8 * 7;
        const int e142 = e141 + 1;
        const int e134 = a79 * 3;
        const double p9_0 = position[e134];
        const int e136 = i8 * 3;
        const int a81 = send_mult[e136];
        const double e139 = a81 * e138;
        const double e140 = p9_0 + e139;
        send_buffer[e142] = e140;
        const int e150 = i8 * 7;
        const int e151 = e150 + 2;
        const int e143 = a79 * 3;
        const int e144 = e143 + 1;
        const double p10_1 = position[e144];
        const int e145 = i8 * 3;
        const int e146 = e145 + 1;
        const int a83 = send_mult[e146];
        const double e148 = a83 * e147;
        const double e149 = p10_1 + e148;
        send_buffer[e151] = e149;
        const int e159 = i8 * 7;
        const int e160 = e159 + 3;
        const int e152 = a79 * 3;
        const int e153 = e152 + 2;
        const double p11_2 = position[e153];
        const int e154 = i8 * 3;
        const int e155 = e154 + 2;
        const int a85 = send_mult[e155];
        const double e157 = a85 * e156;
        const double e158 = p11_2 + e157;
        send_buffer[e160] = e158;
        const int e163 = i8 * 7;
        const int e164 = e163 + 4;
        const int e161 = a79 * 3;
        const double p12_0 = velocity[e161];
        send_buffer[e164] = p12_0;
        const int e167 = i8 * 7;
        const int e168 = e167 + 5;
        const int e165 = a79 * 3;
        const int e166 = e165 + 1;
        const double p13_1 = velocity[e166];
        send_buffer[e168] = p13_1;
        const int e171 = i8 * 7;
        const int e172 = e171 + 6;
        const int e169 = a79 * 3;
        const int e170 = e169 + 2;
        const double p14_2 = velocity[e170];
        send_buffer[e172] = p14_2;
    }
}
__global__ void remove_exchanged_particles_pt2_kernel0(int range_start, int nsend_all, int *exchg_copy_to, int *send_map, double *mass, double *position, double *velocity) {
    const int i10 = blockIdx.x * blockDim.x + threadIdx.x + range_start;
    if((i10 < nsend_all)) {
        const int a94 = exchg_copy_to[i10];
        const bool e179 = a94 > 0;
        if(e179) {
            const int a95 = send_map[i10];
            const double p16 = mass[a94];
            mass[a95] = p16;
            const int e180 = a95 * 3;
            const int e182 = a94 * 3;
            const double p18_0 = position[e182];
            position[e180] = p18_0;
            const int e184 = a95 * 3;
            const int e185 = e184 + 1;
            const int e186 = a94 * 3;
            const int e187 = e186 + 1;
            const double p20_1 = position[e187];
            position[e185] = p20_1;
            const int e188 = a95 * 3;
            const int e189 = e188 + 2;
            const int e190 = a94 * 3;
            const int e191 = e190 + 2;
            const double p22_2 = position[e191];
            position[e189] = p22_2;
            const int e192 = a95 * 3;
            const int e194 = a94 * 3;
            const double p24_0 = velocity[e194];
            velocity[e192] = p24_0;
            const int e196 = a95 * 3;
            const int e197 = e196 + 1;
            const int e198 = a94 * 3;
            const int e199 = e198 + 1;
            const double p26_1 = velocity[e199];
            velocity[e197] = p26_1;
            const int e200 = a95 * 3;
            const int e201 = e200 + 2;
            const int e202 = a94 * 3;
            const int e203 = e202 + 2;
            const double p28_2 = velocity[e203];
            velocity[e201] = p28_2;
        }
    }
}
__global__ void unpack_ghost_particles0_0_1_2_kernel0(int range_start, int nlocal, int *recv_offsets, int *nrecv, double *recv_buffer, double *mass, double *position, double *velocity) {
    const int i11 = blockIdx.x * blockDim.x + threadIdx.x + range_start;
    if((i11 < (recv_offsets[0] + (nrecv[0] + nrecv[1])))) {
        const int e208 = nlocal + i11;
        const int e209 = i11 * 7;
        const double a99 = recv_buffer[e209];
        mass[e208] = a99;
        const int e211 = nlocal + i11;
        const int e212 = e211 * 3;
        const int e214 = i11 * 7;
        const int e215 = e214 + 1;
        const double a100 = recv_buffer[e215];
        position[e212] = a100;
        const int e216 = nlocal + i11;
        const int e217 = e216 * 3;
        const int e218 = e217 + 1;
        const int e219 = i11 * 7;
        const int e220 = e219 + 2;
        const double a101 = recv_buffer[e220];
        position[e218] = a101;
        const int e221 = nlocal + i11;
        const int e222 = e221 * 3;
        const int e223 = e222 + 2;
        const int e224 = i11 * 7;
        const int e225 = e224 + 3;
        const double a102 = recv_buffer[e225];
        position[e223] = a102;
        const int e226 = nlocal + i11;
        const int e227 = e226 * 3;
        const int e229 = i11 * 7;
        const int e230 = e229 + 4;
        const double a103 = recv_buffer[e230];
        velocity[e227] = a103;
        const int e231 = nlocal + i11;
        const int e232 = e231 * 3;
        const int e233 = e232 + 1;
        const int e234 = i11 * 7;
        const int e235 = e234 + 5;
        const double a104 = recv_buffer[e235];
        velocity[e233] = a104;
        const int e236 = nlocal + i11;
        const int e237 = e236 * 3;
        const int e238 = e237 + 2;
        const int e239 = i11 * 7;
        const int e240 = e239 + 6;
        const double a105 = recv_buffer[e240];
        velocity[e238] = a105;
    }
}
__global__ void pack_ghost_particles1_0_1_2_kernel0(int range_start, double grid0_d0_max, double grid0_d0_min, double grid0_d1_max, double grid0_d1_min, double grid0_d2_max, double grid0_d2_min, int *send_offsets, int *nsend, double *send_buffer, int *send_map, int *send_mult, double *mass, double *position, double *velocity, double e285, double e294, double e303) {
    const int i15 = blockIdx.x * blockDim.x + threadIdx.x + range_start;
    if((i15 < (send_offsets[2] + (nsend[2] + nsend[3])))) {
        const int e279 = i15 * 7;
        const int a146 = send_map[i15];
        const double p38 = mass[a146];
        send_buffer[e279] = p38;
        const int e288 = i15 * 7;
        const int e289 = e288 + 1;
        const int e281 = a146 * 3;
        const double p39_0 = position[e281];
        const int e283 = i15 * 3;
        const int a148 = send_mult[e283];
        const double e286 = a148 * e285;
        const double e287 = p39_0 + e286;
        send_buffer[e289] = e287;
        const int e297 = i15 * 7;
        const int e298 = e297 + 2;
        const int e290 = a146 * 3;
        const int e291 = e290 + 1;
        const double p40_1 = position[e291];
        const int e292 = i15 * 3;
        const int e293 = e292 + 1;
        const int a150 = send_mult[e293];
        const double e295 = a150 * e294;
        const double e296 = p40_1 + e295;
        send_buffer[e298] = e296;
        const int e306 = i15 * 7;
        const int e307 = e306 + 3;
        const int e299 = a146 * 3;
        const int e300 = e299 + 2;
        const double p41_2 = position[e300];
        const int e301 = i15 * 3;
        const int e302 = e301 + 2;
        const int a152 = send_mult[e302];
        const double e304 = a152 * e303;
        const double e305 = p41_2 + e304;
        send_buffer[e307] = e305;
        const int e310 = i15 * 7;
        const int e311 = e310 + 4;
        const int e308 = a146 * 3;
        const double p42_0 = velocity[e308];
        send_buffer[e311] = p42_0;
        const int e314 = i15 * 7;
        const int e315 = e314 + 5;
        const int e312 = a146 * 3;
        const int e313 = e312 + 1;
        const double p43_1 = velocity[e313];
        send_buffer[e315] = p43_1;
        const int e318 = i15 * 7;
        const int e319 = e318 + 6;
        const int e316 = a146 * 3;
        const int e317 = e316 + 2;
        const double p44_2 = velocity[e317];
        send_buffer[e319] = p44_2;
    }
}
__global__ void unpack_ghost_particles1_0_1_2_kernel0(int range_start, int nlocal, int *recv_offsets, int *nrecv, double *recv_buffer, double *mass, double *position, double *velocity) {
    const int i18 = blockIdx.x * blockDim.x + threadIdx.x + range_start;
    if((i18 < (recv_offsets[2] + (nrecv[2] + nrecv[3])))) {
        const int e355 = nlocal + i18;
        const int e356 = i18 * 7;
        const double a166 = recv_buffer[e356];
        mass[e355] = a166;
        const int e358 = nlocal + i18;
        const int e359 = e358 * 3;
        const int e361 = i18 * 7;
        const int e362 = e361 + 1;
        const double a167 = recv_buffer[e362];
        position[e359] = a167;
        const int e363 = nlocal + i18;
        const int e364 = e363 * 3;
        const int e365 = e364 + 1;
        const int e366 = i18 * 7;
        const int e367 = e366 + 2;
        const double a168 = recv_buffer[e367];
        position[e365] = a168;
        const int e368 = nlocal + i18;
        const int e369 = e368 * 3;
        const int e370 = e369 + 2;
        const int e371 = i18 * 7;
        const int e372 = e371 + 3;
        const double a169 = recv_buffer[e372];
        position[e370] = a169;
        const int e373 = nlocal + i18;
        const int e374 = e373 * 3;
        const int e376 = i18 * 7;
        const int e377 = e376 + 4;
        const double a170 = recv_buffer[e377];
        velocity[e374] = a170;
        const int e378 = nlocal + i18;
        const int e379 = e378 * 3;
        const int e380 = e379 + 1;
        const int e381 = i18 * 7;
        const int e382 = e381 + 5;
        const double a171 = recv_buffer[e382];
        velocity[e380] = a171;
        const int e383 = nlocal + i18;
        const int e384 = e383 * 3;
        const int e385 = e384 + 2;
        const int e386 = i18 * 7;
        const int e387 = e386 + 6;
        const double a172 = recv_buffer[e387];
        velocity[e385] = a172;
    }
}
__global__ void pack_ghost_particles2_0_1_2_kernel0(int range_start, double grid0_d0_max, double grid0_d0_min, double grid0_d1_max, double grid0_d1_min, double grid0_d2_max, double grid0_d2_min, int *send_offsets, int *nsend, double *send_buffer, int *send_map, int *send_mult, double *mass, double *position, double *velocity, double e436, double e445, double e454) {
    const int i22 = blockIdx.x * blockDim.x + threadIdx.x + range_start;
    if((i22 < (send_offsets[4] + (nsend[4] + nsend[5])))) {
        const int e430 = i22 * 7;
        const int a217 = send_map[i22];
        const double p68 = mass[a217];
        send_buffer[e430] = p68;
        const int e439 = i22 * 7;
        const int e440 = e439 + 1;
        const int e432 = a217 * 3;
        const double p69_0 = position[e432];
        const int e434 = i22 * 3;
        const int a219 = send_mult[e434];
        const double e437 = a219 * e436;
        const double e438 = p69_0 + e437;
        send_buffer[e440] = e438;
        const int e448 = i22 * 7;
        const int e449 = e448 + 2;
        const int e441 = a217 * 3;
        const int e442 = e441 + 1;
        const double p70_1 = position[e442];
        const int e443 = i22 * 3;
        const int e444 = e443 + 1;
        const int a221 = send_mult[e444];
        const double e446 = a221 * e445;
        const double e447 = p70_1 + e446;
        send_buffer[e449] = e447;
        const int e457 = i22 * 7;
        const int e458 = e457 + 3;
        const int e450 = a217 * 3;
        const int e451 = e450 + 2;
        const double p71_2 = position[e451];
        const int e452 = i22 * 3;
        const int e453 = e452 + 2;
        const int a223 = send_mult[e453];
        const double e455 = a223 * e454;
        const double e456 = p71_2 + e455;
        send_buffer[e458] = e456;
        const int e461 = i22 * 7;
        const int e462 = e461 + 4;
        const int e459 = a217 * 3;
        const double p72_0 = velocity[e459];
        send_buffer[e462] = p72_0;
        const int e465 = i22 * 7;
        const int e466 = e465 + 5;
        const int e463 = a217 * 3;
        const int e464 = e463 + 1;
        const double p73_1 = velocity[e464];
        send_buffer[e466] = p73_1;
        const int e469 = i22 * 7;
        const int e470 = e469 + 6;
        const int e467 = a217 * 3;
        const int e468 = e467 + 2;
        const double p74_2 = velocity[e468];
        send_buffer[e470] = p74_2;
    }
}
__global__ void unpack_ghost_particles2_0_1_2_kernel0(int range_start, int nlocal, int *recv_offsets, int *nrecv, double *recv_buffer, double *mass, double *position, double *velocity) {
    const int i25 = blockIdx.x * blockDim.x + threadIdx.x + range_start;
    if((i25 < (recv_offsets[4] + (nrecv[4] + nrecv[5])))) {
        const int e506 = nlocal + i25;
        const int e507 = i25 * 7;
        const double a237 = recv_buffer[e507];
        mass[e506] = a237;
        const int e509 = nlocal + i25;
        const int e510 = e509 * 3;
        const int e512 = i25 * 7;
        const int e513 = e512 + 1;
        const double a238 = recv_buffer[e513];
        position[e510] = a238;
        const int e514 = nlocal + i25;
        const int e515 = e514 * 3;
        const int e516 = e515 + 1;
        const int e517 = i25 * 7;
        const int e518 = e517 + 2;
        const double a239 = recv_buffer[e518];
        position[e516] = a239;
        const int e519 = nlocal + i25;
        const int e520 = e519 * 3;
        const int e521 = e520 + 2;
        const int e522 = i25 * 7;
        const int e523 = e522 + 3;
        const double a240 = recv_buffer[e523];
        position[e521] = a240;
        const int e524 = nlocal + i25;
        const int e525 = e524 * 3;
        const int e527 = i25 * 7;
        const int e528 = e527 + 4;
        const double a241 = recv_buffer[e528];
        velocity[e525] = a241;
        const int e529 = nlocal + i25;
        const int e530 = e529 * 3;
        const int e531 = e530 + 1;
        const int e532 = i25 * 7;
        const int e533 = e532 + 5;
        const double a242 = recv_buffer[e533];
        velocity[e531] = a242;
        const int e534 = nlocal + i25;
        const int e535 = e534 * 3;
        const int e536 = e535 + 2;
        const int e537 = i25 * 7;
        const int e538 = e537 + 6;
        const double a243 = recv_buffer[e538];
        velocity[e536] = a243;
    }
}
__global__ void pack_ghost_particles0_0_1_kernel0(int range_start, double grid0_d0_max, double grid0_d0_min, double grid0_d1_max, double grid0_d1_min, double grid0_d2_max, double grid0_d2_min, int *send_offsets, int *nsend, double *send_buffer, int *send_map, int *send_mult, double *mass, double *position, double e579, double e588, double e597) {
    const int i28 = blockIdx.x * blockDim.x + threadIdx.x + range_start;
    if((i28 < (send_offsets[0] + (nsend[0] + nsend[1])))) {
        const int e573 = i28 * 4;
        const int a277 = send_map[i28];
        const double p98 = mass[a277];
        send_buffer[e573] = p98;
        const int e582 = i28 * 4;
        const int e583 = e582 + 1;
        const int e575 = a277 * 3;
        const double p99_0 = position[e575];
        const int e577 = i28 * 3;
        const int a279 = send_mult[e577];
        const double e580 = a279 * e579;
        const double e581 = p99_0 + e580;
        send_buffer[e583] = e581;
        const int e591 = i28 * 4;
        const int e592 = e591 + 2;
        const int e584 = a277 * 3;
        const int e585 = e584 + 1;
        const double p100_1 = position[e585];
        const int e586 = i28 * 3;
        const int e587 = e586 + 1;
        const int a281 = send_mult[e587];
        const double e589 = a281 * e588;
        const double e590 = p100_1 + e589;
        send_buffer[e592] = e590;
        const int e600 = i28 * 4;
        const int e601 = e600 + 3;
        const int e593 = a277 * 3;
        const int e594 = e593 + 2;
        const double p101_2 = position[e594];
        const int e595 = i28 * 3;
        const int e596 = e595 + 2;
        const int a283 = send_mult[e596];
        const double e598 = a283 * e597;
        const double e599 = p101_2 + e598;
        send_buffer[e601] = e599;
    }
}
__global__ void unpack_ghost_particles0_0_1_kernel0(int range_start, int nlocal, int *recv_offsets, int *nrecv, double *recv_buffer, double *mass, double *position) {
    const int i29 = blockIdx.x * blockDim.x + threadIdx.x + range_start;
    if((i29 < (recv_offsets[0] + (nrecv[0] + nrecv[1])))) {
        const int e605 = nlocal + i29;
        const int e606 = i29 * 4;
        const double a288 = recv_buffer[e606];
        mass[e605] = a288;
        const int e608 = nlocal + i29;
        const int e609 = e608 * 3;
        const int e611 = i29 * 4;
        const int e612 = e611 + 1;
        const double a289 = recv_buffer[e612];
        position[e609] = a289;
        const int e613 = nlocal + i29;
        const int e614 = e613 * 3;
        const int e615 = e614 + 1;
        const int e616 = i29 * 4;
        const int e617 = e616 + 2;
        const double a290 = recv_buffer[e617];
        position[e615] = a290;
        const int e618 = nlocal + i29;
        const int e619 = e618 * 3;
        const int e620 = e619 + 2;
        const int e621 = i29 * 4;
        const int e622 = e621 + 3;
        const double a291 = recv_buffer[e622];
        position[e620] = a291;
    }
}
__global__ void pack_ghost_particles1_0_1_kernel0(int range_start, double grid0_d0_max, double grid0_d0_min, double grid0_d1_max, double grid0_d1_min, double grid0_d2_max, double grid0_d2_min, int *send_offsets, int *nsend, double *send_buffer, int *send_map, int *send_mult, double *mass, double *position, double e664, double e673, double e682) {
    const int i32 = blockIdx.x * blockDim.x + threadIdx.x + range_start;
    if((i32 < (send_offsets[2] + (nsend[2] + nsend[3])))) {
        const int e658 = i32 * 4;
        const int a327 = send_map[i32];
        const double p108 = mass[a327];
        send_buffer[e658] = p108;
        const int e667 = i32 * 4;
        const int e668 = e667 + 1;
        const int e660 = a327 * 3;
        const double p109_0 = position[e660];
        const int e662 = i32 * 3;
        const int a329 = send_mult[e662];
        const double e665 = a329 * e664;
        const double e666 = p109_0 + e665;
        send_buffer[e668] = e666;
        const int e676 = i32 * 4;
        const int e677 = e676 + 2;
        const int e669 = a327 * 3;
        const int e670 = e669 + 1;
        const double p110_1 = position[e670];
        const int e671 = i32 * 3;
        const int e672 = e671 + 1;
        const int a331 = send_mult[e672];
        const double e674 = a331 * e673;
        const double e675 = p110_1 + e674;
        send_buffer[e677] = e675;
        const int e685 = i32 * 4;
        const int e686 = e685 + 3;
        const int e678 = a327 * 3;
        const int e679 = e678 + 2;
        const double p111_2 = position[e679];
        const int e680 = i32 * 3;
        const int e681 = e680 + 2;
        const int a333 = send_mult[e681];
        const double e683 = a333 * e682;
        const double e684 = p111_2 + e683;
        send_buffer[e686] = e684;
    }
}
__global__ void unpack_ghost_particles1_0_1_kernel0(int range_start, int nlocal, int *recv_offsets, int *nrecv, double *recv_buffer, double *mass, double *position) {
    const int i33 = blockIdx.x * blockDim.x + threadIdx.x + range_start;
    if((i33 < (recv_offsets[2] + (nrecv[2] + nrecv[3])))) {
        const int e690 = nlocal + i33;
        const int e691 = i33 * 4;
        const double a338 = recv_buffer[e691];
        mass[e690] = a338;
        const int e693 = nlocal + i33;
        const int e694 = e693 * 3;
        const int e696 = i33 * 4;
        const int e697 = e696 + 1;
        const double a339 = recv_buffer[e697];
        position[e694] = a339;
        const int e698 = nlocal + i33;
        const int e699 = e698 * 3;
        const int e700 = e699 + 1;
        const int e701 = i33 * 4;
        const int e702 = e701 + 2;
        const double a340 = recv_buffer[e702];
        position[e700] = a340;
        const int e703 = nlocal + i33;
        const int e704 = e703 * 3;
        const int e705 = e704 + 2;
        const int e706 = i33 * 4;
        const int e707 = e706 + 3;
        const double a341 = recv_buffer[e707];
        position[e705] = a341;
    }
}
__global__ void pack_ghost_particles2_0_1_kernel0(int range_start, double grid0_d0_max, double grid0_d0_min, double grid0_d1_max, double grid0_d1_min, double grid0_d2_max, double grid0_d2_min, int *send_offsets, int *nsend, double *send_buffer, int *send_map, int *send_mult, double *mass, double *position, double e753, double e762, double e771) {
    const int i36 = blockIdx.x * blockDim.x + threadIdx.x + range_start;
    if((i36 < (send_offsets[4] + (nsend[4] + nsend[5])))) {
        const int e747 = i36 * 4;
        const int a381 = send_map[i36];
        const double p118 = mass[a381];
        send_buffer[e747] = p118;
        const int e756 = i36 * 4;
        const int e757 = e756 + 1;
        const int e749 = a381 * 3;
        const double p119_0 = position[e749];
        const int e751 = i36 * 3;
        const int a383 = send_mult[e751];
        const double e754 = a383 * e753;
        const double e755 = p119_0 + e754;
        send_buffer[e757] = e755;
        const int e765 = i36 * 4;
        const int e766 = e765 + 2;
        const int e758 = a381 * 3;
        const int e759 = e758 + 1;
        const double p120_1 = position[e759];
        const int e760 = i36 * 3;
        const int e761 = e760 + 1;
        const int a385 = send_mult[e761];
        const double e763 = a385 * e762;
        const double e764 = p120_1 + e763;
        send_buffer[e766] = e764;
        const int e774 = i36 * 4;
        const int e775 = e774 + 3;
        const int e767 = a381 * 3;
        const int e768 = e767 + 2;
        const double p121_2 = position[e768];
        const int e769 = i36 * 3;
        const int e770 = e769 + 2;
        const int a387 = send_mult[e770];
        const double e772 = a387 * e771;
        const double e773 = p121_2 + e772;
        send_buffer[e775] = e773;
    }
}
__global__ void unpack_ghost_particles2_0_1_kernel0(int range_start, int nlocal, int *recv_offsets, int *nrecv, double *recv_buffer, double *mass, double *position) {
    const int i37 = blockIdx.x * blockDim.x + threadIdx.x + range_start;
    if((i37 < (recv_offsets[4] + (nrecv[4] + nrecv[5])))) {
        const int e779 = nlocal + i37;
        const int e780 = i37 * 4;
        const double a392 = recv_buffer[e780];
        mass[e779] = a392;
        const int e782 = nlocal + i37;
        const int e783 = e782 * 3;
        const int e785 = i37 * 4;
        const int e786 = e785 + 1;
        const double a393 = recv_buffer[e786];
        position[e783] = a393;
        const int e787 = nlocal + i37;
        const int e788 = e787 * 3;
        const int e789 = e788 + 1;
        const int e790 = i37 * 4;
        const int e791 = e790 + 2;
        const double a394 = recv_buffer[e791];
        position[e789] = a394;
        const int e792 = nlocal + i37;
        const int e793 = e792 * 3;
        const int e794 = e793 + 2;
        const int e795 = i37 * 4;
        const int e796 = e795 + 3;
        const double a395 = recv_buffer[e796];
        position[e794] = a395;
    }
}
__global__ void pack_ghost_particles0_1_kernel0(int range_start, double grid0_d0_max, double grid0_d0_min, double grid0_d1_max, double grid0_d1_min, double grid0_d2_max, double grid0_d2_min, int *send_offsets, int *nsend, double *send_buffer, int *send_map, int *send_mult, double *position, double e804, double e813, double e822) {
    const int i38 = blockIdx.x * blockDim.x + threadIdx.x + range_start;
    if((i38 < (send_offsets[0] + (nsend[0] + nsend[1])))) {
        const int e807 = i38 * 3;
        const int a399 = send_map[i38];
        const int e800 = a399 * 3;
        const double p126_0 = position[e800];
        const int e802 = i38 * 3;
        const int a400 = send_mult[e802];
        const double e805 = a400 * e804;
        const double e806 = p126_0 + e805;
        send_buffer[e807] = e806;
        const int e816 = i38 * 3;
        const int e817 = e816 + 1;
        const int e809 = a399 * 3;
        const int e810 = e809 + 1;
        const double p127_1 = position[e810];
        const int e811 = i38 * 3;
        const int e812 = e811 + 1;
        const int a402 = send_mult[e812];
        const double e814 = a402 * e813;
        const double e815 = p127_1 + e814;
        send_buffer[e817] = e815;
        const int e825 = i38 * 3;
        const int e826 = e825 + 2;
        const int e818 = a399 * 3;
        const int e819 = e818 + 2;
        const double p128_2 = position[e819];
        const int e820 = i38 * 3;
        const int e821 = e820 + 2;
        const int a404 = send_mult[e821];
        const double e823 = a404 * e822;
        const double e824 = p128_2 + e823;
        send_buffer[e826] = e824;
    }
}
__global__ void unpack_ghost_particles0_1_kernel0(int range_start, int nlocal, int *recv_offsets, int *nrecv, double *recv_buffer, double *position) {
    const int i39 = blockIdx.x * blockDim.x + threadIdx.x + range_start;
    if((i39 < (recv_offsets[0] + (nrecv[0] + nrecv[1])))) {
        const int e830 = nlocal + i39;
        const int e831 = e830 * 3;
        const int e833 = i39 * 3;
        const double a409 = recv_buffer[e833];
        position[e831] = a409;
        const int e835 = nlocal + i39;
        const int e836 = e835 * 3;
        const int e837 = e836 + 1;
        const int e838 = i39 * 3;
        const int e839 = e838 + 1;
        const double a410 = recv_buffer[e839];
        position[e837] = a410;
        const int e840 = nlocal + i39;
        const int e841 = e840 * 3;
        const int e842 = e841 + 2;
        const int e843 = i39 * 3;
        const int e844 = e843 + 2;
        const double a411 = recv_buffer[e844];
        position[e842] = a411;
    }
}
__global__ void pack_ghost_particles1_1_kernel0(int range_start, double grid0_d0_max, double grid0_d0_min, double grid0_d1_max, double grid0_d1_min, double grid0_d2_max, double grid0_d2_min, int *send_offsets, int *nsend, double *send_buffer, int *send_map, int *send_mult, double *position, double e852, double e861, double e870) {
    const int i40 = blockIdx.x * blockDim.x + threadIdx.x + range_start;
    if((i40 < (send_offsets[2] + (nsend[2] + nsend[3])))) {
        const int e855 = i40 * 3;
        const int a415 = send_map[i40];
        const int e848 = a415 * 3;
        const double p132_0 = position[e848];
        const int e850 = i40 * 3;
        const int a416 = send_mult[e850];
        const double e853 = a416 * e852;
        const double e854 = p132_0 + e853;
        send_buffer[e855] = e854;
        const int e864 = i40 * 3;
        const int e865 = e864 + 1;
        const int e857 = a415 * 3;
        const int e858 = e857 + 1;
        const double p133_1 = position[e858];
        const int e859 = i40 * 3;
        const int e860 = e859 + 1;
        const int a418 = send_mult[e860];
        const double e862 = a418 * e861;
        const double e863 = p133_1 + e862;
        send_buffer[e865] = e863;
        const int e873 = i40 * 3;
        const int e874 = e873 + 2;
        const int e866 = a415 * 3;
        const int e867 = e866 + 2;
        const double p134_2 = position[e867];
        const int e868 = i40 * 3;
        const int e869 = e868 + 2;
        const int a420 = send_mult[e869];
        const double e871 = a420 * e870;
        const double e872 = p134_2 + e871;
        send_buffer[e874] = e872;
    }
}
__global__ void unpack_ghost_particles1_1_kernel0(int range_start, int nlocal, int *recv_offsets, int *nrecv, double *recv_buffer, double *position) {
    const int i41 = blockIdx.x * blockDim.x + threadIdx.x + range_start;
    if((i41 < (recv_offsets[2] + (nrecv[2] + nrecv[3])))) {
        const int e878 = nlocal + i41;
        const int e879 = e878 * 3;
        const int e881 = i41 * 3;
        const double a425 = recv_buffer[e881];
        position[e879] = a425;
        const int e883 = nlocal + i41;
        const int e884 = e883 * 3;
        const int e885 = e884 + 1;
        const int e886 = i41 * 3;
        const int e887 = e886 + 1;
        const double a426 = recv_buffer[e887];
        position[e885] = a426;
        const int e888 = nlocal + i41;
        const int e889 = e888 * 3;
        const int e890 = e889 + 2;
        const int e891 = i41 * 3;
        const int e892 = e891 + 2;
        const double a427 = recv_buffer[e892];
        position[e890] = a427;
    }
}
__global__ void pack_ghost_particles2_1_kernel0(int range_start, double grid0_d0_max, double grid0_d0_min, double grid0_d1_max, double grid0_d1_min, double grid0_d2_max, double grid0_d2_min, int *send_offsets, int *nsend, double *send_buffer, int *send_map, int *send_mult, double *position, double e900, double e909, double e918) {
    const int i42 = blockIdx.x * blockDim.x + threadIdx.x + range_start;
    if((i42 < (send_offsets[4] + (nsend[4] + nsend[5])))) {
        const int e903 = i42 * 3;
        const int a431 = send_map[i42];
        const int e896 = a431 * 3;
        const double p138_0 = position[e896];
        const int e898 = i42 * 3;
        const int a432 = send_mult[e898];
        const double e901 = a432 * e900;
        const double e902 = p138_0 + e901;
        send_buffer[e903] = e902;
        const int e912 = i42 * 3;
        const int e913 = e912 + 1;
        const int e905 = a431 * 3;
        const int e906 = e905 + 1;
        const double p139_1 = position[e906];
        const int e907 = i42 * 3;
        const int e908 = e907 + 1;
        const int a434 = send_mult[e908];
        const double e910 = a434 * e909;
        const double e911 = p139_1 + e910;
        send_buffer[e913] = e911;
        const int e921 = i42 * 3;
        const int e922 = e921 + 2;
        const int e914 = a431 * 3;
        const int e915 = e914 + 2;
        const double p140_2 = position[e915];
        const int e916 = i42 * 3;
        const int e917 = e916 + 2;
        const int a436 = send_mult[e917];
        const double e919 = a436 * e918;
        const double e920 = p140_2 + e919;
        send_buffer[e922] = e920;
    }
}
__global__ void unpack_ghost_particles2_1_kernel0(int range_start, int nlocal, int *recv_offsets, int *nrecv, double *recv_buffer, double *position) {
    const int i43 = blockIdx.x * blockDim.x + threadIdx.x + range_start;
    if((i43 < (recv_offsets[4] + (nrecv[4] + nrecv[5])))) {
        const int e926 = nlocal + i43;
        const int e927 = e926 * 3;
        const int e929 = i43 * 3;
        const double a441 = recv_buffer[e929];
        position[e927] = a441;
        const int e931 = nlocal + i43;
        const int e932 = e931 * 3;
        const int e933 = e932 + 1;
        const int e934 = i43 * 3;
        const int e935 = e934 + 1;
        const double a442 = recv_buffer[e935];
        position[e933] = a442;
        const int e936 = nlocal + i43;
        const int e937 = e936 * 3;
        const int e938 = e937 + 2;
        const int e939 = i43 * 3;
        const int e940 = e939 + 2;
        const double a443 = recv_buffer[e940];
        position[e938] = a443;
    }
}
__global__ void build_cell_lists_kernel0(int range_start, int ncells, int *cell_sizes) {
    const int i44 = blockIdx.x * blockDim.x + threadIdx.x + range_start;
    if((i44 < ncells)) {
        cell_sizes[i44] = 0;
    }
}
__global__ void build_cell_lists_kernel1(int range_start, int nlocal, int nghost, double grid0_d0_min, double grid0_d1_min, double grid0_d2_min, int ncells, int cell_capacity, int *dim_cells, int *particle_cell, int *cell_particles, int *cell_sizes, int *resizes, double *position, int a446, int a445) {
    const int i45 = blockIdx.x * blockDim.x + threadIdx.x + range_start;
    if((i45 < (nlocal + nghost))) {
        const int e942 = i45 * 3;
        const double p144_0 = position[e942];
        const double e944 = p144_0 - grid0_d0_min;
        const double e945 = e944 / 2.8;
        const int e954 = (int)(e945) * a445;
        const int e946 = i45 * 3;
        const int e947 = e946 + 1;
        const double p145_1 = position[e947];
        const double e948 = p145_1 - grid0_d1_min;
        const double e949 = e948 / 2.8;
        const int e955 = e954 + (int)(e949);
        const int e956 = e955 * a446;
        const int e950 = i45 * 3;
        const int e951 = e950 + 2;
        const double p146_2 = position[e951];
        const double e952 = p146_2 - grid0_d2_min;
        const double e953 = e952 / 2.8;
        const int e957 = e956 + (int)(e953);
        const bool e958 = e957 >= 0;
        const bool e959 = e957 <= ncells;
        const bool e960 = e958 && e959;
        if(e960) {
            particle_cell[i45] = e957;
            const int atm_add12 = pairs::atomic_add_resize_check(&(cell_sizes[e957]), 1, &(resizes[0]), cell_capacity);
            const int e961 = e957 * cell_capacity;
            const int e962 = e961 + atm_add12;
            cell_particles[e962] = i45;
        }
    }
}
__global__ void neighbor_lists_build_kernel0(int range_start, int nlocal, int *numneighs) {
    const int i46 = blockIdx.x * blockDim.x + threadIdx.x + range_start;
    if((i46 < nlocal)) {
        numneighs[i46] = 0;
    }
}
__global__ void neighbor_lists_build_kernel1(int range_start, int nlocal, int ncells, int cell_capacity, int neighborlist_capacity, int nstencil, int *particle_cell, int *stencil, int *cell_particles, int *neighborlists, int *numneighs, int *resizes, int *cell_sizes, double *position) {
    const int i50 = blockIdx.x * blockDim.x + threadIdx.x + range_start;
    if((i50 < nlocal)) {
        const int a456 = particle_cell[i50];
        for(int i51 = 0; i51 < nstencil; i51++) {
            const int a457 = stencil[i51];
            const int e1006 = a456 + a457;
            const bool e1007 = e1006 >= 0;
            const bool e1008 = e1006 <= ncells;
            const bool e1009 = e1007 && e1008;
            if(e1009) {
                const int a458 = cell_sizes[e1006];
                const int e1010 = e1006 * cell_capacity;
                const int e1018 = i50 * 3;
                const int e1027 = i50 * 3;
                const int e1028 = e1027 + 1;
                const int e1037 = i50 * 3;
                const int e1038 = e1037 + 2;
                const double p150_0 = position[e1018];
                const double p150_1 = position[e1028];
                const double p150_2 = position[e1038];
                const int e963 = i50 * neighborlist_capacity;
                for(int i52 = 0; i52 < a458; i52++) {
                    const int e1011 = e1010 + i52;
                    const int a459 = cell_particles[e1011];
                    const bool e1012 = a459 != i50;
                    if(e1012) {
                        const int e1020 = a459 * 3;
                        const int e1029 = a459 * 3;
                        const int e1030 = e1029 + 1;
                        const int e1039 = a459 * 3;
                        const int e1040 = e1039 + 2;
                        const double p151_0 = position[e1020];
                        const double p151_1 = position[e1030];
                        const double p151_2 = position[e1040];
                        const double e1013_0 = p150_0 - p151_0;
                        const double e1013_1 = p150_1 - p151_1;
                        const double e1013_2 = p150_2 - p151_2;
                        const double e1022 = e1013_0 * e1013_0;
                        const double e1031 = e1013_1 * e1013_1;
                        const double e1032 = e1022 + e1031;
                        const double e1041 = e1013_2 * e1013_2;
                        const double e1042 = e1032 + e1041;
                        const bool e1043 = e1042 < 2.8;
                        if(e1043) {
                            const int a451 = numneighs[i50];
                            const int e964 = e963 + a451;
                            neighborlists[e964] = a459;
                            const int e965 = a451 + 1;
                            const int e1153 = e965 + 1;
                            const bool e1154 = e1153 >= neighborlist_capacity;
                            if(e1154) {
                                resizes[0] = e965;
                            } else {
                                numneighs[i50] = e965;
                            }
                        }
                    }
                }
            }
        }
    }
}
__global__ void reset_volatile_properties_kernel0(int range_start, int nlocal, double *force) {
    const int i47 = blockIdx.x * blockDim.x + threadIdx.x + range_start;
    if((i47 < nlocal)) {
        const int e966 = i47 * 3;
        const int e968 = i47 * 3;
        const int e969 = e968 + 1;
        const int e970 = i47 * 3;
        const int e971 = e970 + 2;
        force[e966] = 0.0;
        force[e969] = 0.0;
        force[e971] = 0.0;
    }
}
__global__ void lj_kernel0(int range_start, int nlocal, int neighborlist_capacity, int *neighborlists, int *numneighs, double *position, double *force) {
    const int i48 = blockIdx.x * blockDim.x + threadIdx.x + range_start;
    if((i48 < nlocal)) {
        const int a454 = numneighs[i48];
        const int e979 = i48 * 3;
        const int e988 = i48 * 3;
        const int e989 = e988 + 1;
        const int e998 = i48 * 3;
        const int e999 = e998 + 2;
        const double p148_0 = position[e979];
        const double p148_1 = position[e989];
        const double p148_2 = position[e999];
        const int e972 = i48 * neighborlist_capacity;
        const int e14 = i48 * 3;
        const int e18 = i48 * 3;
        const int e19 = e18 + 1;
        const int e22 = i48 * 3;
        const int e23 = e22 + 2;
        for(int i49 = 0; i49 < a454; i49++) {
            const int e973 = e972 + i49;
            const int a455 = neighborlists[e973];
            const int e981 = a455 * 3;
            const int e990 = a455 * 3;
            const int e991 = e990 + 1;
            const int e1000 = a455 * 3;
            const int e1001 = e1000 + 2;
            const double p149_0 = position[e981];
            const double p149_1 = position[e991];
            const double p149_2 = position[e1001];
            const double e974_0 = p148_0 - p149_0;
            const double e974_1 = p148_1 - p149_1;
            const double e974_2 = p148_2 - p149_2;
            const double e983 = e974_0 * e974_0;
            const double e992 = e974_1 * e974_1;
            const double e993 = e983 + e992;
            const double e1002 = e974_2 * e974_2;
            const double e1003 = e993 + e1002;
            const bool e1004 = e1003 < 2.5;
            if(e1004) {
                const double p0_0 = force[e14];
                const double p0_1 = force[e19];
                const double p0_2 = force[e23];
                const double e1 = 1.0 / e1003;
                const double e2 = e1 * e1;
                const double e3 = e2 * e1;
                const double e1044 = 48.0 * e3;
                const double e7 = e3 - 0.5;
                const double e1045 = e1044 * e7;
                const double e1046 = e1045 * e1;
                const double e9_0 = e974_0 * e1046;
                const double e9_1 = e974_1 * e1046;
                const double e9_2 = e974_2 * e1046;
                const double e11_0 = p0_0 + e9_0;
                const double e11_1 = p0_1 + e9_1;
                const double e11_2 = p0_2 + e9_2;
                force[e14] = e11_0;
                force[e19] = e11_1;
                force[e23] = e11_2;
            }
        }
    }
}
__global__ void euler_kernel0(int range_start, int nlocal, double *velocity, double *force, double *mass, double *position) {
    const int i0 = blockIdx.x * blockDim.x + threadIdx.x + range_start;
    if((i0 < nlocal)) {
        const int e31 = i0 * 3;
        const int e37 = i0 * 3;
        const int e38 = e37 + 1;
        const int e43 = i0 * 3;
        const int e44 = e43 + 2;
        const double p1_0 = velocity[e31];
        const double p1_1 = velocity[e38];
        const double p1_2 = velocity[e44];
        const int e29 = i0 * 3;
        const int e35 = i0 * 3;
        const int e36 = e35 + 1;
        const int e41 = i0 * 3;
        const int e42 = e41 + 2;
        const double p2_0 = force[e29];
        const double p2_1 = force[e36];
        const double p2_2 = force[e42];
        const double e24_0 = 0.005 * p2_0;
        const double e24_1 = 0.005 * p2_1;
        const double e24_2 = 0.005 * p2_2;
        const double p3 = mass[i0];
        const double e25_0 = e24_0 / p3;
        const double e25_1 = e24_1 / p3;
        const double e25_2 = e24_2 / p3;
        const double e26_0 = p1_0 + e25_0;
        const double e26_1 = p1_1 + e25_1;
        const double e26_2 = p1_2 + e25_2;
        velocity[e31] = e26_0;
        velocity[e38] = e26_1;
        velocity[e44] = e26_2;
        const int e51 = i0 * 3;
        const int e57 = i0 * 3;
        const int e58 = e57 + 1;
        const int e63 = i0 * 3;
        const int e64 = e63 + 2;
        const double p4_0 = position[e51];
        const double p4_1 = position[e58];
        const double p4_2 = position[e64];
        const int e49 = i0 * 3;
        const int e55 = i0 * 3;
        const int e56 = e55 + 1;
        const int e61 = i0 * 3;
        const int e62 = e61 + 2;
        const double p5_0 = velocity[e49];
        const double p5_1 = velocity[e56];
        const double p5_2 = velocity[e62];
        const double e45_0 = 0.005 * p5_0;
        const double e45_1 = 0.005 * p5_1;
        const double e45_2 = 0.005 * p5_2;
        const double e46_0 = p4_0 + e45_0;
        const double e46_1 = p4_1 + e45_1;
        const double e46_2 = p4_2 + e45_2;
        position[e51] = e46_0;
        position[e58] = e46_1;
        position[e64] = e46_2;
    }
}
void lj(PairsSimulation *pairs, int neighborlist_capacity, int nlocal, int *numneighs, int *neighborlists, double *position, double *force) {
    PAIRS_DEBUG("lj\n");
    const int e1427 = nlocal - 0;
    const int e1428 = e1427 + 32;
    const int e1429 = e1428 - 1;
    const int e1430 = e1429 / 32;
    if(e1430 > 0 && 32 > 0) {
        lj_kernel0<<<e1430, 32>>>(0, nlocal, neighborlist_capacity, neighborlists, numneighs, position, force);
        pairs->sync();
    }
}
void euler(PairsSimulation *pairs, int nlocal, double *velocity, double *force, double *mass, double *position) {
    PAIRS_DEBUG("euler\n");
    const int e1432 = nlocal - 0;
    const int e1433 = e1432 + 32;
    const int e1434 = e1433 - 1;
    const int e1435 = e1434 / 32;
    if(e1435 > 0 && 32 > 0) {
        euler_kernel0<<<e1435, 32>>>(0, nlocal, velocity, force, mass, position);
        pairs->sync();
    }
}
void build_cell_lists_stencil(PairsSimulation *pairs, double grid0_d0_max, double grid0_d0_min, double grid0_d1_max, double grid0_d1_min, double grid0_d2_max, double grid0_d2_min, int ncells_capacity, int *ncells, int *nstencil, int *dim_cells, int *resizes, int *stencil) {
    PAIRS_DEBUG("build_cell_lists_stencil\n");
    const double e83 = grid0_d0_max - grid0_d0_min;
    const double e84 = e83 / 2.8;
    const int e85 = ceil(e84) + 2;
    dim_cells[0] = e85;
    const double e87 = grid0_d1_max - grid0_d1_min;
    const double e88 = e87 / 2.8;
    const int e89 = ceil(e88) + 2;
    dim_cells[1] = e89;
    const double e91 = grid0_d2_max - grid0_d2_min;
    const double e92 = e91 / 2.8;
    const int e93 = ceil(e92) + 2;
    dim_cells[2] = e93;
    const int a37 = dim_cells[0];
    const int a39 = dim_cells[1];
    const int e90 = a37 * a39;
    const int a41 = dim_cells[2];
    const int e94 = e90 * a41;
    const int e1047 = e94 + 1;
    const bool e1048 = e1047 >= ncells_capacity;
    if(e1048) {
        resizes[0] = e94;
    } else {
        (*ncells) = e94;
    }
    (*nstencil) = 0;
    for(int i2 = -1; i2 < 2; i2++) {
        for(int i3 = -1; i3 < 2; i3++) {
            const int a42 = dim_cells[0];
            const int e95 = i2 * a42;
            const int e96 = e95 + i3;
            const int a43 = dim_cells[1];
            const int e97 = e96 * a43;
            for(int i4 = -1; i4 < 2; i4++) {
                const int e98 = e97 + i4;
                stencil[(*nstencil)] = e98;
                const int e99 = (*nstencil) + 1;
                (*nstencil) = e99;
            }
        }
    }
}
void determine_exchange_particles0(PairsSimulation *pairs, int nlocal, int nghost, int send_capacity, int *nsend_all, int *nsend, int *nrecv, int *exchg_flag, double *subdom, int *pbc, int *send_map, int *send_mult, int *resizes, double *position) {
    PAIRS_DEBUG("determine_exchange_particles0\n");
    nsend[0] = 0;
    nrecv[0] = 0;
    nsend[1] = 0;
    nrecv[1] = 0;
    for(int i5 = 0; i5 < nlocal; i5++) {
        exchg_flag[i5] = 0;
    }
    const int e101 = nlocal + nghost;
    const double a50 = subdom[0];
    const int a52 = pbc[0];
    for(int i6 = 0; i6 < e101; i6++) {
        const int e102 = i6 * 3;
        const double p6_0 = position[e102];
        const bool e105 = p6_0 < a50;
        if(e105) {
            const int atm_add0 = pairs::host_atomic_add(&((*nsend_all)), 1);
            send_map[atm_add0] = i6;
            exchg_flag[i6] = 1;
            const int e106 = atm_add0 * 3;
            send_mult[e106] = a52;
            const int e108 = atm_add0 * 3;
            const int e109 = e108 + 1;
            send_mult[e109] = 0;
            const int e110 = atm_add0 * 3;
            const int e111 = e110 + 2;
            send_mult[e111] = 0;
            const int a58 = nsend[0];
            const int e112 = a58 + 1;
            const int e1055 = e112 + 1;
            const bool e1056 = e1055 >= send_capacity;
            if(e1056) {
                resizes[0] = e112;
            } else {
                nsend[0] = e112;
            }
        }
    }
    const int e113 = nlocal + nghost;
    const double a59 = subdom[1];
    const int a61 = pbc[1];
    for(int i7 = 0; i7 < e113; i7++) {
        const int e114 = i7 * 3;
        const double p7_0 = position[e114];
        const bool e117 = p7_0 > a59;
        if(e117) {
            const int atm_add1 = pairs::host_atomic_add(&((*nsend_all)), 1);
            send_map[atm_add1] = i7;
            exchg_flag[i7] = 1;
            const int e118 = atm_add1 * 3;
            send_mult[e118] = a61;
            const int e120 = atm_add1 * 3;
            const int e121 = e120 + 1;
            send_mult[e121] = 0;
            const int e122 = atm_add1 * 3;
            const int e123 = e122 + 2;
            send_mult[e123] = 0;
            const int a67 = nsend[1];
            const int e124 = a67 + 1;
            const int e1063 = e124 + 1;
            const bool e1064 = e1063 >= send_capacity;
            if(e1064) {
                resizes[0] = e124;
            } else {
                nsend[1] = e124;
            }
        }
    }
}
void set_communication_offsets0(PairsSimulation *pairs, int *send_offsets, int *recv_offsets, int *nsend, int *nrecv) {
    PAIRS_DEBUG("set_communication_offsets0\n");
    send_offsets[0] = 0;
    recv_offsets[0] = 0;
    const int a70 = nsend[0];
    send_offsets[1] = a70;
    const int a71 = nrecv[0];
    recv_offsets[1] = a71;
}
void pack_ghost_particles0_0_1_2(PairsSimulation *pairs, double grid0_d0_max, double grid0_d0_min, double grid0_d1_max, double grid0_d1_min, double grid0_d2_max, double grid0_d2_min, int *send_map, double *send_buffer, int *send_mult, int *send_offsets, int *h_send_offsets, int *nsend, int *h_nsend, double *mass, double *position, double *velocity) {
    PAIRS_DEBUG("pack_ghost_particles0_0_1_2\n");
    const double e138 = grid0_d0_max - grid0_d0_min;
    const double e147 = grid0_d1_max - grid0_d1_min;
    const double e156 = grid0_d2_max - grid0_d2_min;
    const int e1264 = (h_send_offsets[0] + (h_nsend[0] + h_nsend[1])) - h_send_offsets[0];
    const int e1265 = e1264 + 32;
    const int e1266 = e1265 - 1;
    const int e1267 = e1266 / 32;
    if(e1267 > 0 && 32 > 0) {
        pack_ghost_particles0_0_1_2_kernel0<<<e1267, 32>>>(h_send_offsets[0], grid0_d0_max, grid0_d0_min, grid0_d1_max, grid0_d1_min, grid0_d2_max, grid0_d2_min, send_offsets, nsend, send_buffer, send_map, send_mult, mass, position, velocity, e138, e147, e156);
        pairs->sync();
    }
}
void remove_exchanged_particles_pt1(PairsSimulation *pairs, int nlocal, int nsend_all, int *send_map, int *exchg_flag, int *exchg_copy_to) {
    PAIRS_DEBUG("remove_exchanged_particles_pt1\n");
    int tmp0 = 0;
    const int e173 = nlocal - 1;
    tmp0 = e173;
    const int e174 = nlocal - nsend_all;
    for(int i9 = 0; i9 < nsend_all; i9++) {
        const int a90 = send_map[i9];
        const bool e175 = a90 < e174;
        if(e175) {
            while((exchg_flag[tmp0] == 1)) {
                const int e177 = tmp0 - 1;
                tmp0 = e177;
            }
            exchg_copy_to[i9] = tmp0;
            const int e178 = tmp0 - 1;
            tmp0 = e178;
        } else {
            exchg_copy_to[i9] = -1;
        }
    }
}
void remove_exchanged_particles_pt2(PairsSimulation *pairs, int nsend_all, int *nlocal, int *exchg_copy_to, int *send_map, double *mass, double *position, double *velocity) {
    PAIRS_DEBUG("remove_exchanged_particles_pt2\n");
    const int e1269 = nsend_all - 0;
    const int e1270 = e1269 + 32;
    const int e1271 = e1270 - 1;
    const int e1272 = e1271 / 32;
    if(e1272 > 0 && 32 > 0) {
        remove_exchanged_particles_pt2_kernel0<<<e1272, 32>>>(0, nsend_all, exchg_copy_to, send_map, mass, position, velocity);
        pairs->sync();
    }
    const int e204 = (*nlocal) - nsend_all;
    (*nlocal) = e204;
}
void unpack_ghost_particles0_0_1_2(PairsSimulation *pairs, int nlocal, double *recv_buffer, int *recv_offsets, int *h_recv_offsets, int *nrecv, int *h_nrecv, double *mass, double *position, double *velocity) {
    PAIRS_DEBUG("unpack_ghost_particles0_0_1_2\n");
    const int e1276 = (h_recv_offsets[0] + (h_nrecv[0] + h_nrecv[1])) - h_recv_offsets[0];
    const int e1277 = e1276 + 32;
    const int e1278 = e1277 - 1;
    const int e1279 = e1278 / 32;
    if(e1279 > 0 && 32 > 0) {
        unpack_ghost_particles0_0_1_2_kernel0<<<e1279, 32>>>(h_recv_offsets[0], nlocal, recv_offsets, nrecv, recv_buffer, mass, position, velocity);
        pairs->sync();
    }
}
void change_size_after_exchange0(PairsSimulation *pairs, int particle_capacity, int *nlocal, int *nrecv, int *resizes) {
    PAIRS_DEBUG("change_size_after_exchange0\n");
    const int a106 = nrecv[0];
    const int a107 = nrecv[1];
    const int e242 = a106 + a107;
    const int e243 = (*nlocal) + e242;
    const int e1065 = e243 + 1;
    const bool e1066 = e1065 >= particle_capacity;
    if(e1066) {
        resizes[0] = e243;
    } else {
        (*nlocal) = e243;
    }
}
void determine_exchange_particles1(PairsSimulation *pairs, int nlocal, int nghost, int send_capacity, int *nsend_all, int *nsend, int *nrecv, int *exchg_flag, double *subdom, int *pbc, int *send_map, int *send_mult, int *resizes, double *position) {
    PAIRS_DEBUG("determine_exchange_particles1\n");
    nsend[2] = 0;
    nrecv[2] = 0;
    nsend[3] = 0;
    nrecv[3] = 0;
    for(int i12 = 0; i12 < nlocal; i12++) {
        exchg_flag[i12] = 0;
    }
    const int e244 = nlocal + nghost;
    const double a113 = subdom[2];
    const int a115 = pbc[2];
    for(int i13 = 0; i13 < e244; i13++) {
        const int e245 = i13 * 3;
        const int e246 = e245 + 1;
        const double p36_1 = position[e246];
        const bool e248 = p36_1 < a113;
        if(e248) {
            const int atm_add2 = pairs::host_atomic_add(&((*nsend_all)), 1);
            send_map[atm_add2] = i13;
            exchg_flag[i13] = 1;
            const int e249 = atm_add2 * 3;
            send_mult[e249] = 0;
            const int e251 = atm_add2 * 3;
            const int e252 = e251 + 1;
            send_mult[e252] = a115;
            const int e253 = atm_add2 * 3;
            const int e254 = e253 + 2;
            send_mult[e254] = 0;
            const int a121 = nsend[2];
            const int e255 = a121 + 1;
            const int e1073 = e255 + 1;
            const bool e1074 = e1073 >= send_capacity;
            if(e1074) {
                resizes[0] = e255;
            } else {
                nsend[2] = e255;
            }
        }
    }
    const int e256 = nlocal + nghost;
    const double a122 = subdom[3];
    const int a124 = pbc[3];
    for(int i14 = 0; i14 < e256; i14++) {
        const int e257 = i14 * 3;
        const int e258 = e257 + 1;
        const double p37_1 = position[e258];
        const bool e260 = p37_1 > a122;
        if(e260) {
            const int atm_add3 = pairs::host_atomic_add(&((*nsend_all)), 1);
            send_map[atm_add3] = i14;
            exchg_flag[i14] = 1;
            const int e261 = atm_add3 * 3;
            send_mult[e261] = 0;
            const int e263 = atm_add3 * 3;
            const int e264 = e263 + 1;
            send_mult[e264] = a124;
            const int e265 = atm_add3 * 3;
            const int e266 = e265 + 2;
            send_mult[e266] = 0;
            const int a130 = nsend[3];
            const int e267 = a130 + 1;
            const int e1081 = e267 + 1;
            const bool e1082 = e1081 >= send_capacity;
            if(e1082) {
                resizes[0] = e267;
            } else {
                nsend[3] = e267;
            }
        }
    }
}
void set_communication_offsets1(PairsSimulation *pairs, int *nsend, int *send_offsets, int *nrecv, int *recv_offsets) {
    PAIRS_DEBUG("set_communication_offsets1\n");
    const int a131 = nsend[0];
    const int a133 = nsend[1];
    const int e270 = a131 + a133;
    send_offsets[2] = e270;
    const int a132 = nrecv[0];
    const int a134 = nrecv[1];
    const int e271 = a132 + a134;
    recv_offsets[2] = e271;
    const int a137 = nsend[2];
    const int e272 = e270 + a137;
    send_offsets[3] = e272;
    const int a138 = nrecv[2];
    const int e273 = e271 + a138;
    recv_offsets[3] = e273;
}
void pack_ghost_particles1_0_1_2(PairsSimulation *pairs, double grid0_d0_max, double grid0_d0_min, double grid0_d1_max, double grid0_d1_min, double grid0_d2_max, double grid0_d2_min, int *send_map, double *send_buffer, int *send_mult, int *send_offsets, int *h_send_offsets, int *nsend, int *h_nsend, double *mass, double *position, double *velocity) {
    PAIRS_DEBUG("pack_ghost_particles1_0_1_2\n");
    const double e285 = grid0_d0_max - grid0_d0_min;
    const double e294 = grid0_d1_max - grid0_d1_min;
    const double e303 = grid0_d2_max - grid0_d2_min;
    const int e1283 = (h_send_offsets[2] + (h_nsend[2] + h_nsend[3])) - h_send_offsets[2];
    const int e1284 = e1283 + 32;
    const int e1285 = e1284 - 1;
    const int e1286 = e1285 / 32;
    if(e1286 > 0 && 32 > 0) {
        pack_ghost_particles1_0_1_2_kernel0<<<e1286, 32>>>(h_send_offsets[2], grid0_d0_max, grid0_d0_min, grid0_d1_max, grid0_d1_min, grid0_d2_max, grid0_d2_min, send_offsets, nsend, send_buffer, send_map, send_mult, mass, position, velocity, e285, e294, e303);
        pairs->sync();
    }
}
void unpack_ghost_particles1_0_1_2(PairsSimulation *pairs, int nlocal, double *recv_buffer, int *recv_offsets, int *h_recv_offsets, int *nrecv, int *h_nrecv, double *mass, double *position, double *velocity) {
    PAIRS_DEBUG("unpack_ghost_particles1_0_1_2\n");
    const int e1294 = (h_recv_offsets[2] + (h_nrecv[2] + h_nrecv[3])) - h_recv_offsets[2];
    const int e1295 = e1294 + 32;
    const int e1296 = e1295 - 1;
    const int e1297 = e1296 / 32;
    if(e1297 > 0 && 32 > 0) {
        unpack_ghost_particles1_0_1_2_kernel0<<<e1297, 32>>>(h_recv_offsets[2], nlocal, recv_offsets, nrecv, recv_buffer, mass, position, velocity);
        pairs->sync();
    }
}
void change_size_after_exchange1(PairsSimulation *pairs, int particle_capacity, int *nlocal, int *nrecv, int *resizes) {
    PAIRS_DEBUG("change_size_after_exchange1\n");
    const int a173 = nrecv[2];
    const int a174 = nrecv[3];
    const int e389 = a173 + a174;
    const int e390 = (*nlocal) + e389;
    const int e1083 = e390 + 1;
    const bool e1084 = e1083 >= particle_capacity;
    if(e1084) {
        resizes[0] = e390;
    } else {
        (*nlocal) = e390;
    }
}
void determine_exchange_particles2(PairsSimulation *pairs, int nlocal, int nghost, int send_capacity, int *nsend_all, int *nsend, int *nrecv, int *exchg_flag, double *subdom, int *pbc, int *send_map, int *send_mult, int *resizes, double *position) {
    PAIRS_DEBUG("determine_exchange_particles2\n");
    nsend[4] = 0;
    nrecv[4] = 0;
    nsend[5] = 0;
    nrecv[5] = 0;
    for(int i19 = 0; i19 < nlocal; i19++) {
        exchg_flag[i19] = 0;
    }
    const int e391 = nlocal + nghost;
    const double a180 = subdom[4];
    const int a182 = pbc[4];
    for(int i20 = 0; i20 < e391; i20++) {
        const int e392 = i20 * 3;
        const int e393 = e392 + 2;
        const double p66_2 = position[e393];
        const bool e395 = p66_2 < a180;
        if(e395) {
            const int atm_add4 = pairs::host_atomic_add(&((*nsend_all)), 1);
            send_map[atm_add4] = i20;
            exchg_flag[i20] = 1;
            const int e396 = atm_add4 * 3;
            send_mult[e396] = 0;
            const int e398 = atm_add4 * 3;
            const int e399 = e398 + 1;
            send_mult[e399] = 0;
            const int e400 = atm_add4 * 3;
            const int e401 = e400 + 2;
            send_mult[e401] = a182;
            const int a188 = nsend[4];
            const int e402 = a188 + 1;
            const int e1091 = e402 + 1;
            const bool e1092 = e1091 >= send_capacity;
            if(e1092) {
                resizes[0] = e402;
            } else {
                nsend[4] = e402;
            }
        }
    }
    const int e403 = nlocal + nghost;
    const double a189 = subdom[5];
    const int a191 = pbc[5];
    for(int i21 = 0; i21 < e403; i21++) {
        const int e404 = i21 * 3;
        const int e405 = e404 + 2;
        const double p67_2 = position[e405];
        const bool e407 = p67_2 > a189;
        if(e407) {
            const int atm_add5 = pairs::host_atomic_add(&((*nsend_all)), 1);
            send_map[atm_add5] = i21;
            exchg_flag[i21] = 1;
            const int e408 = atm_add5 * 3;
            send_mult[e408] = 0;
            const int e410 = atm_add5 * 3;
            const int e411 = e410 + 1;
            send_mult[e411] = 0;
            const int e412 = atm_add5 * 3;
            const int e413 = e412 + 2;
            send_mult[e413] = a191;
            const int a197 = nsend[5];
            const int e414 = a197 + 1;
            const int e1099 = e414 + 1;
            const bool e1100 = e1099 >= send_capacity;
            if(e1100) {
                resizes[0] = e414;
            } else {
                nsend[5] = e414;
            }
        }
    }
}
void set_communication_offsets2(PairsSimulation *pairs, int *nsend, int *send_offsets, int *nrecv, int *recv_offsets) {
    PAIRS_DEBUG("set_communication_offsets2\n");
    const int a198 = nsend[0];
    const int a200 = nsend[1];
    const int e417 = a198 + a200;
    const int a202 = nsend[2];
    const int e419 = e417 + a202;
    const int a204 = nsend[3];
    const int e421 = e419 + a204;
    send_offsets[4] = e421;
    const int a199 = nrecv[0];
    const int a201 = nrecv[1];
    const int e418 = a199 + a201;
    const int a203 = nrecv[2];
    const int e420 = e418 + a203;
    const int a205 = nrecv[3];
    const int e422 = e420 + a205;
    recv_offsets[4] = e422;
    const int a208 = nsend[4];
    const int e423 = e421 + a208;
    send_offsets[5] = e423;
    const int a209 = nrecv[4];
    const int e424 = e422 + a209;
    recv_offsets[5] = e424;
}
void pack_ghost_particles2_0_1_2(PairsSimulation *pairs, double grid0_d0_max, double grid0_d0_min, double grid0_d1_max, double grid0_d1_min, double grid0_d2_max, double grid0_d2_min, int *send_map, double *send_buffer, int *send_mult, int *send_offsets, int *h_send_offsets, int *nsend, int *h_nsend, double *mass, double *position, double *velocity) {
    PAIRS_DEBUG("pack_ghost_particles2_0_1_2\n");
    const double e436 = grid0_d0_max - grid0_d0_min;
    const double e445 = grid0_d1_max - grid0_d1_min;
    const double e454 = grid0_d2_max - grid0_d2_min;
    const int e1301 = (h_send_offsets[4] + (h_nsend[4] + h_nsend[5])) - h_send_offsets[4];
    const int e1302 = e1301 + 32;
    const int e1303 = e1302 - 1;
    const int e1304 = e1303 / 32;
    if(e1304 > 0 && 32 > 0) {
        pack_ghost_particles2_0_1_2_kernel0<<<e1304, 32>>>(h_send_offsets[4], grid0_d0_max, grid0_d0_min, grid0_d1_max, grid0_d1_min, grid0_d2_max, grid0_d2_min, send_offsets, nsend, send_buffer, send_map, send_mult, mass, position, velocity, e436, e445, e454);
        pairs->sync();
    }
}
void unpack_ghost_particles2_0_1_2(PairsSimulation *pairs, int nlocal, double *recv_buffer, int *recv_offsets, int *h_recv_offsets, int *nrecv, int *h_nrecv, double *mass, double *position, double *velocity) {
    PAIRS_DEBUG("unpack_ghost_particles2_0_1_2\n");
    const int e1312 = (h_recv_offsets[4] + (h_nrecv[4] + h_nrecv[5])) - h_recv_offsets[4];
    const int e1313 = e1312 + 32;
    const int e1314 = e1313 - 1;
    const int e1315 = e1314 / 32;
    if(e1315 > 0 && 32 > 0) {
        unpack_ghost_particles2_0_1_2_kernel0<<<e1315, 32>>>(h_recv_offsets[4], nlocal, recv_offsets, nrecv, recv_buffer, mass, position, velocity);
        pairs->sync();
    }
}
void change_size_after_exchange2(PairsSimulation *pairs, int particle_capacity, int *nlocal, int *nrecv, int *resizes) {
    PAIRS_DEBUG("change_size_after_exchange2\n");
    const int a244 = nrecv[4];
    const int a245 = nrecv[5];
    const int e540 = a244 + a245;
    const int e541 = (*nlocal) + e540;
    const int e1101 = e541 + 1;
    const bool e1102 = e1101 >= particle_capacity;
    if(e1102) {
        resizes[0] = e541;
    } else {
        (*nlocal) = e541;
    }
}
void determine_ghost_particles0(PairsSimulation *pairs, int nlocal, int nghost, int send_capacity, int *nsend_all, int *nsend, int *nrecv, double *subdom, int *pbc, int *send_map, int *send_mult, int *resizes, double *position) {
    PAIRS_DEBUG("determine_ghost_particles0\n");
    nsend[0] = 0;
    nrecv[0] = 0;
    nsend[1] = 0;
    nrecv[1] = 0;
    const int e542 = nlocal + nghost;
    const double a250 = subdom[0];
    const double e545 = a250 + 2.8;
    const int a252 = pbc[0];
    for(int i26 = 0; i26 < e542; i26++) {
        const int e543 = i26 * 3;
        const double p96_0 = position[e543];
        const bool e546 = p96_0 < e545;
        if(e546) {
            const int atm_add6 = pairs::host_atomic_add(&((*nsend_all)), 1);
            send_map[atm_add6] = i26;
            const int e547 = atm_add6 * 3;
            send_mult[e547] = a252;
            const int e549 = atm_add6 * 3;
            const int e550 = e549 + 1;
            send_mult[e550] = 0;
            const int e551 = atm_add6 * 3;
            const int e552 = e551 + 2;
            send_mult[e552] = 0;
            const int a257 = nsend[0];
            const int e553 = a257 + 1;
            const int e1109 = e553 + 1;
            const bool e1110 = e1109 >= send_capacity;
            if(e1110) {
                resizes[0] = e553;
            } else {
                nsend[0] = e553;
            }
        }
    }
    const int e554 = nlocal + nghost;
    const double a258 = subdom[1];
    const double e557 = a258 - 2.8;
    const int a260 = pbc[1];
    for(int i27 = 0; i27 < e554; i27++) {
        const int e555 = i27 * 3;
        const double p97_0 = position[e555];
        const bool e558 = p97_0 > e557;
        if(e558) {
            const int atm_add7 = pairs::host_atomic_add(&((*nsend_all)), 1);
            send_map[atm_add7] = i27;
            const int e559 = atm_add7 * 3;
            send_mult[e559] = a260;
            const int e561 = atm_add7 * 3;
            const int e562 = e561 + 1;
            send_mult[e562] = 0;
            const int e563 = atm_add7 * 3;
            const int e564 = e563 + 2;
            send_mult[e564] = 0;
            const int a265 = nsend[1];
            const int e565 = a265 + 1;
            const int e1117 = e565 + 1;
            const bool e1118 = e1117 >= send_capacity;
            if(e1118) {
                resizes[0] = e565;
            } else {
                nsend[1] = e565;
            }
        }
    }
}
void pack_ghost_particles0_0_1(PairsSimulation *pairs, double grid0_d0_max, double grid0_d0_min, double grid0_d1_max, double grid0_d1_min, double grid0_d2_max, double grid0_d2_min, int *send_map, double *send_buffer, int *send_mult, int *send_offsets, int *h_send_offsets, int *nsend, int *h_nsend, double *mass, double *position) {
    PAIRS_DEBUG("pack_ghost_particles0_0_1\n");
    const double e579 = grid0_d0_max - grid0_d0_min;
    const double e588 = grid0_d1_max - grid0_d1_min;
    const double e597 = grid0_d2_max - grid0_d2_min;
    const int e1319 = (h_send_offsets[0] + (h_nsend[0] + h_nsend[1])) - h_send_offsets[0];
    const int e1320 = e1319 + 32;
    const int e1321 = e1320 - 1;
    const int e1322 = e1321 / 32;
    if(e1322 > 0 && 32 > 0) {
        pack_ghost_particles0_0_1_kernel0<<<e1322, 32>>>(h_send_offsets[0], grid0_d0_max, grid0_d0_min, grid0_d1_max, grid0_d1_min, grid0_d2_max, grid0_d2_min, send_offsets, nsend, send_buffer, send_map, send_mult, mass, position, e579, e588, e597);
        pairs->sync();
    }
}
void unpack_ghost_particles0_0_1(PairsSimulation *pairs, int nlocal, double *recv_buffer, int *recv_offsets, int *h_recv_offsets, int *nrecv, int *h_nrecv, double *mass, double *position) {
    PAIRS_DEBUG("unpack_ghost_particles0_0_1\n");
    const int e1326 = (h_recv_offsets[0] + (h_nrecv[0] + h_nrecv[1])) - h_recv_offsets[0];
    const int e1327 = e1326 + 32;
    const int e1328 = e1327 - 1;
    const int e1329 = e1328 / 32;
    if(e1329 > 0 && 32 > 0) {
        unpack_ghost_particles0_0_1_kernel0<<<e1329, 32>>>(h_recv_offsets[0], nlocal, recv_offsets, nrecv, recv_buffer, mass, position);
        pairs->sync();
    }
}
void determine_ghost_particles1(PairsSimulation *pairs, int nlocal, int nghost, int send_capacity, int *nsend_all, int *nsend, int *nrecv, double *subdom, int *pbc, int *send_map, int *send_mult, int *resizes, double *position) {
    PAIRS_DEBUG("determine_ghost_particles1\n");
    nsend[2] = 0;
    nrecv[2] = 0;
    nsend[3] = 0;
    nrecv[3] = 0;
    const int e623 = nlocal + nghost;
    const double a296 = subdom[2];
    const double e626 = a296 + 2.8;
    const int a298 = pbc[2];
    for(int i30 = 0; i30 < e623; i30++) {
        const int e624 = i30 * 3;
        const int e625 = e624 + 1;
        const double p106_1 = position[e625];
        const bool e627 = p106_1 < e626;
        if(e627) {
            const int atm_add8 = pairs::host_atomic_add(&((*nsend_all)), 1);
            send_map[atm_add8] = i30;
            const int e628 = atm_add8 * 3;
            send_mult[e628] = 0;
            const int e630 = atm_add8 * 3;
            const int e631 = e630 + 1;
            send_mult[e631] = a298;
            const int e632 = atm_add8 * 3;
            const int e633 = e632 + 2;
            send_mult[e633] = 0;
            const int a303 = nsend[2];
            const int e634 = a303 + 1;
            const int e1125 = e634 + 1;
            const bool e1126 = e1125 >= send_capacity;
            if(e1126) {
                resizes[0] = e634;
            } else {
                nsend[2] = e634;
            }
        }
    }
    const int e635 = nlocal + nghost;
    const double a304 = subdom[3];
    const double e638 = a304 - 2.8;
    const int a306 = pbc[3];
    for(int i31 = 0; i31 < e635; i31++) {
        const int e636 = i31 * 3;
        const int e637 = e636 + 1;
        const double p107_1 = position[e637];
        const bool e639 = p107_1 > e638;
        if(e639) {
            const int atm_add9 = pairs::host_atomic_add(&((*nsend_all)), 1);
            send_map[atm_add9] = i31;
            const int e640 = atm_add9 * 3;
            send_mult[e640] = 0;
            const int e642 = atm_add9 * 3;
            const int e643 = e642 + 1;
            send_mult[e643] = a306;
            const int e644 = atm_add9 * 3;
            const int e645 = e644 + 2;
            send_mult[e645] = 0;
            const int a311 = nsend[3];
            const int e646 = a311 + 1;
            const int e1133 = e646 + 1;
            const bool e1134 = e1133 >= send_capacity;
            if(e1134) {
                resizes[0] = e646;
            } else {
                nsend[3] = e646;
            }
        }
    }
}
void pack_ghost_particles1_0_1(PairsSimulation *pairs, double grid0_d0_max, double grid0_d0_min, double grid0_d1_max, double grid0_d1_min, double grid0_d2_max, double grid0_d2_min, int *send_map, double *send_buffer, int *send_mult, int *send_offsets, int *h_send_offsets, int *nsend, int *h_nsend, double *mass, double *position) {
    PAIRS_DEBUG("pack_ghost_particles1_0_1\n");
    const double e664 = grid0_d0_max - grid0_d0_min;
    const double e673 = grid0_d1_max - grid0_d1_min;
    const double e682 = grid0_d2_max - grid0_d2_min;
    const int e1333 = (h_send_offsets[2] + (h_nsend[2] + h_nsend[3])) - h_send_offsets[2];
    const int e1334 = e1333 + 32;
    const int e1335 = e1334 - 1;
    const int e1336 = e1335 / 32;
    if(e1336 > 0 && 32 > 0) {
        pack_ghost_particles1_0_1_kernel0<<<e1336, 32>>>(h_send_offsets[2], grid0_d0_max, grid0_d0_min, grid0_d1_max, grid0_d1_min, grid0_d2_max, grid0_d2_min, send_offsets, nsend, send_buffer, send_map, send_mult, mass, position, e664, e673, e682);
        pairs->sync();
    }
}
void unpack_ghost_particles1_0_1(PairsSimulation *pairs, int nlocal, double *recv_buffer, int *recv_offsets, int *h_recv_offsets, int *nrecv, int *h_nrecv, double *mass, double *position) {
    PAIRS_DEBUG("unpack_ghost_particles1_0_1\n");
    const int e1340 = (h_recv_offsets[2] + (h_nrecv[2] + h_nrecv[3])) - h_recv_offsets[2];
    const int e1341 = e1340 + 32;
    const int e1342 = e1341 - 1;
    const int e1343 = e1342 / 32;
    if(e1343 > 0 && 32 > 0) {
        unpack_ghost_particles1_0_1_kernel0<<<e1343, 32>>>(h_recv_offsets[2], nlocal, recv_offsets, nrecv, recv_buffer, mass, position);
        pairs->sync();
    }
}
void determine_ghost_particles2(PairsSimulation *pairs, int nlocal, int nghost, int send_capacity, int *nsend_all, int *nsend, int *nrecv, double *subdom, int *pbc, int *send_map, int *send_mult, int *resizes, double *position) {
    PAIRS_DEBUG("determine_ghost_particles2\n");
    nsend[4] = 0;
    nrecv[4] = 0;
    nsend[5] = 0;
    nrecv[5] = 0;
    const int e708 = nlocal + nghost;
    const double a346 = subdom[4];
    const double e711 = a346 + 2.8;
    const int a348 = pbc[4];
    for(int i34 = 0; i34 < e708; i34++) {
        const int e709 = i34 * 3;
        const int e710 = e709 + 2;
        const double p116_2 = position[e710];
        const bool e712 = p116_2 < e711;
        if(e712) {
            const int atm_add10 = pairs::host_atomic_add(&((*nsend_all)), 1);
            send_map[atm_add10] = i34;
            const int e713 = atm_add10 * 3;
            send_mult[e713] = 0;
            const int e715 = atm_add10 * 3;
            const int e716 = e715 + 1;
            send_mult[e716] = 0;
            const int e717 = atm_add10 * 3;
            const int e718 = e717 + 2;
            send_mult[e718] = a348;
            const int a353 = nsend[4];
            const int e719 = a353 + 1;
            const int e1141 = e719 + 1;
            const bool e1142 = e1141 >= send_capacity;
            if(e1142) {
                resizes[0] = e719;
            } else {
                nsend[4] = e719;
            }
        }
    }
    const int e720 = nlocal + nghost;
    const double a354 = subdom[5];
    const double e723 = a354 - 2.8;
    const int a356 = pbc[5];
    for(int i35 = 0; i35 < e720; i35++) {
        const int e721 = i35 * 3;
        const int e722 = e721 + 2;
        const double p117_2 = position[e722];
        const bool e724 = p117_2 > e723;
        if(e724) {
            const int atm_add11 = pairs::host_atomic_add(&((*nsend_all)), 1);
            send_map[atm_add11] = i35;
            const int e725 = atm_add11 * 3;
            send_mult[e725] = 0;
            const int e727 = atm_add11 * 3;
            const int e728 = e727 + 1;
            send_mult[e728] = 0;
            const int e729 = atm_add11 * 3;
            const int e730 = e729 + 2;
            send_mult[e730] = a356;
            const int a361 = nsend[5];
            const int e731 = a361 + 1;
            const int e1149 = e731 + 1;
            const bool e1150 = e1149 >= send_capacity;
            if(e1150) {
                resizes[0] = e731;
            } else {
                nsend[5] = e731;
            }
        }
    }
}
void pack_ghost_particles2_0_1(PairsSimulation *pairs, double grid0_d0_max, double grid0_d0_min, double grid0_d1_max, double grid0_d1_min, double grid0_d2_max, double grid0_d2_min, int *send_map, double *send_buffer, int *send_mult, int *send_offsets, int *h_send_offsets, int *nsend, int *h_nsend, double *mass, double *position) {
    PAIRS_DEBUG("pack_ghost_particles2_0_1\n");
    const double e753 = grid0_d0_max - grid0_d0_min;
    const double e762 = grid0_d1_max - grid0_d1_min;
    const double e771 = grid0_d2_max - grid0_d2_min;
    const int e1347 = (h_send_offsets[4] + (h_nsend[4] + h_nsend[5])) - h_send_offsets[4];
    const int e1348 = e1347 + 32;
    const int e1349 = e1348 - 1;
    const int e1350 = e1349 / 32;
    if(e1350 > 0 && 32 > 0) {
        pack_ghost_particles2_0_1_kernel0<<<e1350, 32>>>(h_send_offsets[4], grid0_d0_max, grid0_d0_min, grid0_d1_max, grid0_d1_min, grid0_d2_max, grid0_d2_min, send_offsets, nsend, send_buffer, send_map, send_mult, mass, position, e753, e762, e771);
        pairs->sync();
    }
}
void unpack_ghost_particles2_0_1(PairsSimulation *pairs, int nlocal, double *recv_buffer, int *recv_offsets, int *h_recv_offsets, int *nrecv, int *h_nrecv, double *mass, double *position) {
    PAIRS_DEBUG("unpack_ghost_particles2_0_1\n");
    const int e1354 = (h_recv_offsets[4] + (h_nrecv[4] + h_nrecv[5])) - h_recv_offsets[4];
    const int e1355 = e1354 + 32;
    const int e1356 = e1355 - 1;
    const int e1357 = e1356 / 32;
    if(e1357 > 0 && 32 > 0) {
        unpack_ghost_particles2_0_1_kernel0<<<e1357, 32>>>(h_recv_offsets[4], nlocal, recv_offsets, nrecv, recv_buffer, mass, position);
        pairs->sync();
    }
}
void pack_ghost_particles0_1(PairsSimulation *pairs, double grid0_d0_max, double grid0_d0_min, double grid0_d1_max, double grid0_d1_min, double grid0_d2_max, double grid0_d2_min, int *send_map, int *send_mult, double *send_buffer, int *send_offsets, int *h_send_offsets, int *nsend, int *h_nsend, double *position) {
    PAIRS_DEBUG("pack_ghost_particles0_1\n");
    const double e804 = grid0_d0_max - grid0_d0_min;
    const double e813 = grid0_d1_max - grid0_d1_min;
    const double e822 = grid0_d2_max - grid0_d2_min;
    const int e1361 = (h_send_offsets[0] + (h_nsend[0] + h_nsend[1])) - h_send_offsets[0];
    const int e1362 = e1361 + 32;
    const int e1363 = e1362 - 1;
    const int e1364 = e1363 / 32;
    if(e1364 > 0 && 32 > 0) {
        pack_ghost_particles0_1_kernel0<<<e1364, 32>>>(h_send_offsets[0], grid0_d0_max, grid0_d0_min, grid0_d1_max, grid0_d1_min, grid0_d2_max, grid0_d2_min, send_offsets, nsend, send_buffer, send_map, send_mult, position, e804, e813, e822);
        pairs->sync();
    }
}
void unpack_ghost_particles0_1(PairsSimulation *pairs, int nlocal, double *recv_buffer, int *recv_offsets, int *h_recv_offsets, int *nrecv, int *h_nrecv, double *position) {
    PAIRS_DEBUG("unpack_ghost_particles0_1\n");
    const int e1368 = (h_recv_offsets[0] + (h_nrecv[0] + h_nrecv[1])) - h_recv_offsets[0];
    const int e1369 = e1368 + 32;
    const int e1370 = e1369 - 1;
    const int e1371 = e1370 / 32;
    if(e1371 > 0 && 32 > 0) {
        unpack_ghost_particles0_1_kernel0<<<e1371, 32>>>(h_recv_offsets[0], nlocal, recv_offsets, nrecv, recv_buffer, position);
        pairs->sync();
    }
}
void pack_ghost_particles1_1(PairsSimulation *pairs, double grid0_d0_max, double grid0_d0_min, double grid0_d1_max, double grid0_d1_min, double grid0_d2_max, double grid0_d2_min, int *send_map, int *send_mult, double *send_buffer, int *send_offsets, int *h_send_offsets, int *nsend, int *h_nsend, double *position) {
    PAIRS_DEBUG("pack_ghost_particles1_1\n");
    const double e852 = grid0_d0_max - grid0_d0_min;
    const double e861 = grid0_d1_max - grid0_d1_min;
    const double e870 = grid0_d2_max - grid0_d2_min;
    const int e1375 = (h_send_offsets[2] + (h_nsend[2] + h_nsend[3])) - h_send_offsets[2];
    const int e1376 = e1375 + 32;
    const int e1377 = e1376 - 1;
    const int e1378 = e1377 / 32;
    if(e1378 > 0 && 32 > 0) {
        pack_ghost_particles1_1_kernel0<<<e1378, 32>>>(h_send_offsets[2], grid0_d0_max, grid0_d0_min, grid0_d1_max, grid0_d1_min, grid0_d2_max, grid0_d2_min, send_offsets, nsend, send_buffer, send_map, send_mult, position, e852, e861, e870);
        pairs->sync();
    }
}
void unpack_ghost_particles1_1(PairsSimulation *pairs, int nlocal, double *recv_buffer, int *recv_offsets, int *h_recv_offsets, int *nrecv, int *h_nrecv, double *position) {
    PAIRS_DEBUG("unpack_ghost_particles1_1\n");
    const int e1382 = (h_recv_offsets[2] + (h_nrecv[2] + h_nrecv[3])) - h_recv_offsets[2];
    const int e1383 = e1382 + 32;
    const int e1384 = e1383 - 1;
    const int e1385 = e1384 / 32;
    if(e1385 > 0 && 32 > 0) {
        unpack_ghost_particles1_1_kernel0<<<e1385, 32>>>(h_recv_offsets[2], nlocal, recv_offsets, nrecv, recv_buffer, position);
        pairs->sync();
    }
}
void pack_ghost_particles2_1(PairsSimulation *pairs, double grid0_d0_max, double grid0_d0_min, double grid0_d1_max, double grid0_d1_min, double grid0_d2_max, double grid0_d2_min, int *send_map, int *send_mult, double *send_buffer, int *send_offsets, int *h_send_offsets, int *nsend, int *h_nsend, double *position) {
    PAIRS_DEBUG("pack_ghost_particles2_1\n");
    const double e900 = grid0_d0_max - grid0_d0_min;
    const double e909 = grid0_d1_max - grid0_d1_min;
    const double e918 = grid0_d2_max - grid0_d2_min;
    const int e1389 = (h_send_offsets[4] + (h_nsend[4] + h_nsend[5])) - h_send_offsets[4];
    const int e1390 = e1389 + 32;
    const int e1391 = e1390 - 1;
    const int e1392 = e1391 / 32;
    if(e1392 > 0 && 32 > 0) {
        pack_ghost_particles2_1_kernel0<<<e1392, 32>>>(h_send_offsets[4], grid0_d0_max, grid0_d0_min, grid0_d1_max, grid0_d1_min, grid0_d2_max, grid0_d2_min, send_offsets, nsend, send_buffer, send_map, send_mult, position, e900, e909, e918);
        pairs->sync();
    }
}
void unpack_ghost_particles2_1(PairsSimulation *pairs, int nlocal, double *recv_buffer, int *recv_offsets, int *h_recv_offsets, int *nrecv, int *h_nrecv, double *position) {
    PAIRS_DEBUG("unpack_ghost_particles2_1\n");
    const int e1396 = (h_recv_offsets[4] + (h_nrecv[4] + h_nrecv[5])) - h_recv_offsets[4];
    const int e1397 = e1396 + 32;
    const int e1398 = e1397 - 1;
    const int e1399 = e1398 / 32;
    if(e1399 > 0 && 32 > 0) {
        unpack_ghost_particles2_1_kernel0<<<e1399, 32>>>(h_recv_offsets[4], nlocal, recv_offsets, nrecv, recv_buffer, position);
        pairs->sync();
    }
}
void build_cell_lists(PairsSimulation *pairs, int ncells, int nlocal, int nghost, double grid0_d0_min, double grid0_d1_min, double grid0_d2_min, int cell_capacity, int *cell_sizes, int *dim_cells, int *particle_cell, int *resizes, int *cell_particles, double *position) {
    PAIRS_DEBUG("build_cell_lists\n");
    const int e1401 = ncells - 0;
    const int e1402 = e1401 + 32;
    const int e1403 = e1402 - 1;
    const int e1404 = e1403 / 32;
    if(e1404 > 0 && 32 > 0) {
        build_cell_lists_kernel0<<<e1404, 32>>>(0, ncells, cell_sizes);
        pairs->sync();
    }
    const int e941 = nlocal + nghost;
    const int a445 = dim_cells[1];
    const int a446 = dim_cells[2];
    const int e1407 = e941 - 0;
    const int e1408 = e1407 + 32;
    const int e1409 = e1408 - 1;
    const int e1410 = e1409 / 32;
    if(e1410 > 0 && 32 > 0) {
        build_cell_lists_kernel1<<<e1410, 32>>>(0, nlocal, nghost, grid0_d0_min, grid0_d1_min, grid0_d2_min, ncells, cell_capacity, dim_cells, particle_cell, cell_particles, cell_sizes, resizes, position, a446, a445);
        pairs->sync();
    }
}
void neighbor_lists_build(PairsSimulation *pairs, int nlocal, int ncells, int cell_capacity, int neighborlist_capacity, int nstencil, int *numneighs, int *particle_cell, int *stencil, int *cell_sizes, int *cell_particles, int *neighborlists, int *resizes, double *position) {
    PAIRS_DEBUG("neighbor_lists_build\n");
    const int e1412 = nlocal - 0;
    const int e1413 = e1412 + 32;
    const int e1414 = e1413 - 1;
    const int e1415 = e1414 / 32;
    if(e1415 > 0 && 32 > 0) {
        neighbor_lists_build_kernel0<<<e1415, 32>>>(0, nlocal, numneighs);
        pairs->sync();
    }
    const int e1417 = nlocal - 0;
    const int e1418 = e1417 + 32;
    const int e1419 = e1418 - 1;
    const int e1420 = e1419 / 32;
    if(e1420 > 0 && 32 > 0) {
        neighbor_lists_build_kernel1<<<e1420, 32>>>(0, nlocal, ncells, cell_capacity, neighborlist_capacity, nstencil, particle_cell, stencil, cell_particles, neighborlists, numneighs, resizes, cell_sizes, position);
        pairs->sync();
    }
}
void reset_volatile_properties(PairsSimulation *pairs, int nlocal, double *force) {
    PAIRS_DEBUG("reset_volatile_properties\n");
    const int e1422 = nlocal - 0;
    const int e1423 = e1422 + 32;
    const int e1424 = e1423 - 1;
    const int e1425 = e1424 / 32;
    if(e1425 > 0 && 32 > 0) {
        reset_volatile_properties_kernel0<<<e1425, 32>>>(0, nlocal, force);
        pairs->sync();
    }
}
int main(int argc, char **argv) {
    PairsSimulation *pairs = new PairsSimulation(4, 24, DimRanges);
    int particle_capacity = 1000000;
    int nlocal = 0;
    int nghost = 0;
    double grid0_d0_min = 0;
    double grid0_d0_max = 0;
    double grid0_d1_min = 0;
    double grid0_d1_max = 0;
    double grid0_d2_min = 0;
    double grid0_d2_max = 0;
    int nstencil = 0;
    int ncells = 1;
    int ncells_capacity = 100;
    int cell_capacity = 20;
    int neighborlist_capacity = 32;
    int nsend_all = 0;
    int send_capacity = 100000;
    int recv_capacity = 100000;
    int elem_capacity = 10;
    int neigh_capacity = 6;
    int *resizes, *d_resizes;
    pairs->addArray(0, "resizes", &resizes, &d_resizes, (sizeof(int) * 3));
    pairs->clearArrayHostFlag(0); // resizes
    pairs->clearArrayDeviceFlag(0); // resizes
    int neighbor_ranks[6];
    pairs->addStaticArray(1, "neighbor_ranks", neighbor_ranks, nullptr, (sizeof(int) * 6));
    pairs->clearArrayHostFlag(0); // resizes
    pairs->clearArrayDeviceFlag(0); // resizes
    int pbc[6];
    pairs->addStaticArray(2, "pbc", pbc, nullptr, (sizeof(int) * 6));
    pairs->clearArrayHostFlag(0); // resizes
    pairs->clearArrayDeviceFlag(0); // resizes
    double subdom[6];
    pairs->addStaticArray(3, "subdom", subdom, nullptr, (sizeof(double) * 6));
    pairs->clearArrayHostFlag(0); // resizes
    pairs->clearArrayDeviceFlag(0); // resizes
    double grid_buffer[6];
    pairs->addStaticArray(4, "grid_buffer", grid_buffer, nullptr, (sizeof(double) * 6));
    pairs->clearArrayHostFlag(0); // resizes
    pairs->clearArrayDeviceFlag(0); // resizes
    int dim_cells[3];
    pairs->addStaticArray(5, "dim_cells", dim_cells, d_dim_cells, (sizeof(int) * 3));
    pairs->clearArrayHostFlag(0); // resizes
    pairs->clearArrayDeviceFlag(0); // resizes
    int *cell_particles, *d_cell_particles;
    pairs->addArray(6, "cell_particles", &cell_particles, &d_cell_particles, (sizeof(int) * (ncells_capacity * cell_capacity)));
    int *cell_sizes, *d_cell_sizes;
    pairs->addArray(7, "cell_sizes", &cell_sizes, &d_cell_sizes, (sizeof(int) * ncells_capacity));
    int *stencil, *d_stencil;
    pairs->addArray(8, "stencil", &stencil, &d_stencil, (sizeof(int) * 27));
    int *particle_cell, *d_particle_cell;
    pairs->addArray(9, "particle_cell", &particle_cell, &d_particle_cell, (sizeof(int) * particle_capacity));
    int *neighborlists, *d_neighborlists;
    pairs->addArray(10, "neighborlists", &neighborlists, &d_neighborlists, (sizeof(int) * (particle_capacity * neighborlist_capacity)));
    int *numneighs, *d_numneighs;
    pairs->addArray(11, "numneighs", &numneighs, &d_numneighs, (sizeof(int) * particle_capacity));
    int *nsend, *d_nsend;
    pairs->addArray(12, "nsend", &nsend, &d_nsend, (sizeof(int) * neigh_capacity));
    int *send_offsets, *d_send_offsets;
    pairs->addArray(13, "send_offsets", &send_offsets, &d_send_offsets, (sizeof(int) * neigh_capacity));
    double *send_buffer, *d_send_buffer;
    pairs->addArray(14, "send_buffer", &send_buffer, &d_send_buffer, (sizeof(double) * (send_capacity * elem_capacity)));
    int *send_map, *d_send_map;
    pairs->addArray(15, "send_map", &send_map, &d_send_map, (sizeof(int) * send_capacity));
    int *exchg_flag;
    pairs->addArray(16, "exchg_flag", &exchg_flag, nullptr, (sizeof(int) * particle_capacity));
    int *exchg_copy_to, *d_exchg_copy_to;
    pairs->addArray(17, "exchg_copy_to", &exchg_copy_to, &d_exchg_copy_to, (sizeof(int) * send_capacity));
    int *send_mult, *d_send_mult;
    pairs->addArray(18, "send_mult", &send_mult, &d_send_mult, (sizeof(int) * (send_capacity * 3)));
    int *nrecv, *d_nrecv;
    pairs->addArray(19, "nrecv", &nrecv, &d_nrecv, (sizeof(int) * neigh_capacity));
    int *recv_offsets, *d_recv_offsets;
    pairs->addArray(20, "recv_offsets", &recv_offsets, &d_recv_offsets, (sizeof(int) * neigh_capacity));
    double *recv_buffer, *d_recv_buffer;
    pairs->addArray(21, "recv_buffer", &recv_buffer, &d_recv_buffer, (sizeof(double) * (recv_capacity * elem_capacity)));
    int *recv_map;
    pairs->addArray(22, "recv_map", &recv_map, nullptr, (sizeof(int) * recv_capacity));
    int *recv_mult;
    pairs->addArray(23, "recv_mult", &recv_mult, nullptr, (sizeof(int) * (recv_capacity * 3)));
    double *mass, *d_mass;
    pairs->addProperty(0, "mass", &mass, &d_mass, Prop_Float, AoS, (0 + particle_capacity));
    double *position, *d_position;
    pairs->addProperty(1, "position", &position, &d_position, Prop_Vector, AoS, (0 + particle_capacity), 3);
    double *velocity, *d_velocity;
    pairs->addProperty(2, "velocity", &velocity, &d_velocity, Prop_Vector, AoS, (0 + particle_capacity), 3);
    double *force, *d_force;
    pairs->addProperty(3, "force", &force, &d_force, Prop_Vector, AoS, (0 + particle_capacity), 3);
    pairs::read_grid_data(pairs, "data/minimd_setup_32x32x32.input", grid_buffer);
    const double a30 = grid_buffer[0];
    grid0_d0_min = a30;
    const double a31 = grid_buffer[1];
    grid0_d0_max = a31;
    const double a32 = grid_buffer[2];
    grid0_d1_min = a32;
    const double a33 = grid_buffer[3];
    grid0_d1_max = a33;
    const double a34 = grid_buffer[4];
    grid0_d2_min = a34;
    const double a35 = grid_buffer[5];
    grid0_d2_max = a35;
    pairs->initDomain(&argc, &argv, grid0_d0_min, grid0_d0_max, grid0_d1_min, grid0_d1_max, grid0_d2_min, grid0_d2_max);
    pairs->fillCommunicationArrays(neighbor_ranks, pbc, subdom);
    const int prop_list_0[] = {0, 1, 2};
    nlocal = pairs::read_particle_data(pairs, "data/minimd_setup_32x32x32.input", prop_list_0, 3);
    resizes[0] = 1;
    while((resizes[0] > 0)) {
        resizes[0] = 0;
        pairs->copyArrayToHost(8); // stencil
        pairs->setArrayHostFlag(8); // stencil
        pairs->clearArrayDeviceFlag(8); // stencil
        build_cell_lists_stencil(pairs, grid0_d0_max, grid0_d0_min, grid0_d1_max, grid0_d1_min, grid0_d2_max, grid0_d2_min, ncells_capacity, &ncells, &nstencil, dim_cells, resizes, stencil);
        const int a482 = resizes[0];
        const bool e1156 = a482 > 0;
        if(e1156) {
            PAIRS_DEBUG("resizes[0] -> ncells_capacity\n");
            const int a483 = resizes[0];
            const int e1157 = a483 * 2;
            ncells_capacity = e1157;
            pairs->reallocArray(6, &cell_particles, &d_cell_particles, (sizeof(int) * (ncells_capacity * cell_capacity)));
            pairs->reallocArray(7, &cell_sizes, &d_cell_sizes, (sizeof(int) * ncells_capacity));
        }
    }
    pairs::vtk_write_data(pairs, "output/test_gpu_local", 0, nlocal, 0);
    const int e100 = nlocal + nghost;
    pairs::vtk_write_data(pairs, "output/test_gpu_ghost", nlocal, e100, 0);
    for(int i1 = 0; i1 < 101; i1++) {
        if(((i1 % 20) == 0)) {
            nsend_all = 0;
            nghost = 0;
            resizes[0] = 1;
            while((resizes[0] > 0)) {
                resizes[0] = 0;
                pairs->copyArrayToHost(15); // send_map
                pairs->copyArrayToHost(16); // exchg_flag
                pairs->copyArrayToHost(18); // send_mult
                pairs->copyArrayToHost(19); // nrecv
                pairs->copyArrayToHost(12); // nsend
                pairs->copyPropertyToHost(1); // position
                pairs->setArrayHostFlag(15); // send_map
                pairs->clearArrayDeviceFlag(15); // send_map
                pairs->setArrayHostFlag(16); // exchg_flag
                pairs->clearArrayDeviceFlag(16); // exchg_flag
                pairs->setArrayHostFlag(18); // send_mult
                pairs->clearArrayDeviceFlag(18); // send_mult
                pairs->setArrayHostFlag(19); // nrecv
                pairs->clearArrayDeviceFlag(19); // nrecv
                pairs->setArrayHostFlag(12); // nsend
                pairs->clearArrayDeviceFlag(12); // nsend
                determine_exchange_particles0(pairs, nlocal, nghost, send_capacity, &nsend_all, nsend, nrecv, exchg_flag, subdom, pbc, send_map, send_mult, resizes, position);
                const int a487 = resizes[0];
                const bool e1162 = a487 > 0;
                if(e1162) {
                    PAIRS_DEBUG("resizes[0] -> send_capacity\n");
                    const int a488 = resizes[0];
                    const int e1163 = a488 * 2;
                    send_capacity = e1163;
                    pairs->reallocArray(14, &send_buffer, &d_send_buffer, (sizeof(double) * (send_capacity * elem_capacity)));
                    pairs->reallocArray(15, &send_map, &d_send_map, (sizeof(int) * send_capacity));
                    pairs->reallocArray(17, &exchg_copy_to, &d_exchg_copy_to, (sizeof(int) * send_capacity));
                    pairs->reallocArray(18, &send_mult, &d_send_mult, (sizeof(int) * (send_capacity * 3)));
                }
            }
            pairs->communicateSizes(0, nsend, nrecv);
            pairs->copyArrayToHost(19); // nrecv
            pairs->copyArrayToHost(20); // recv_offsets
            pairs->copyArrayToHost(12); // nsend
            pairs->copyArrayToHost(13); // send_offsets
            pairs->setArrayHostFlag(20); // recv_offsets
            pairs->clearArrayDeviceFlag(20); // recv_offsets
            pairs->setArrayHostFlag(13); // send_offsets
            pairs->clearArrayDeviceFlag(13); // send_offsets
            set_communication_offsets0(pairs, send_offsets, recv_offsets, nsend, nrecv);
            pairs->copyArrayToDevice(15); // send_map
            pairs->copyArrayToDevice(13); // send_offsets
            pairs->copyArrayToDevice(18); // send_mult
            pairs->copyArrayToDevice(12); // nsend
            pairs->copyArrayToDevice(14); // send_buffer
            pairs->copyPropertyToDevice(2); // velocity
            pairs->copyPropertyToDevice(0); // mass
            pairs->copyPropertyToDevice(1); // position
            pairs->setArrayDeviceFlag(14); // send_buffer
            pairs->clearArrayHostFlag(14); // send_buffer
            pack_ghost_particles0_0_1_2(pairs, grid0_d0_max, grid0_d0_min, grid0_d1_max, grid0_d1_min, grid0_d2_max, grid0_d2_min, d_send_map, d_send_buffer, d_send_mult, d_send_offsets, send_offsets, d_nsend, nsend, d_mass, d_position, d_velocity);
            pairs->copyArrayToHost(15); // send_map
            pairs->copyArrayToHost(16); // exchg_flag
            pairs->copyArrayToHost(17); // exchg_copy_to
            pairs->setArrayHostFlag(17); // exchg_copy_to
            pairs->clearArrayDeviceFlag(17); // exchg_copy_to
            remove_exchanged_particles_pt1(pairs, nlocal, nsend_all, send_map, exchg_flag, exchg_copy_to);
            pairs->copyArrayToDevice(15); // send_map
            pairs->copyArrayToDevice(17); // exchg_copy_to
            pairs->copyPropertyToDevice(2); // velocity
            pairs->copyPropertyToDevice(0); // mass
            pairs->copyPropertyToDevice(1); // position
            pairs->setPropertyDeviceFlag(2); // velocity
            pairs->clearPropertyHostFlag(2); // velocity
            pairs->setPropertyDeviceFlag(0); // mass
            pairs->clearPropertyHostFlag(0); // mass
            pairs->setPropertyDeviceFlag(1); // position
            pairs->clearPropertyHostFlag(1); // position
            remove_exchanged_particles_pt2(pairs, nsend_all, &nlocal, d_exchg_copy_to, d_send_map, d_mass, d_position, d_velocity);
            pairs->communicateData(0, 7, send_buffer, send_offsets, nsend, recv_buffer, recv_offsets, nrecv);
            pairs->copyArrayToDevice(21); // recv_buffer
            pairs->copyArrayToDevice(19); // nrecv
            pairs->copyArrayToDevice(20); // recv_offsets
            pairs->copyPropertyToDevice(2); // velocity
            pairs->copyPropertyToDevice(0); // mass
            pairs->copyPropertyToDevice(1); // position
            pairs->setPropertyDeviceFlag(2); // velocity
            pairs->clearPropertyHostFlag(2); // velocity
            pairs->setPropertyDeviceFlag(0); // mass
            pairs->clearPropertyHostFlag(0); // mass
            pairs->setPropertyDeviceFlag(1); // position
            pairs->clearPropertyHostFlag(1); // position
            unpack_ghost_particles0_0_1_2(pairs, nlocal, d_recv_buffer, d_recv_offsets, recv_offsets, d_nrecv, nrecv, d_mass, d_position, d_velocity);
            resizes[0] = 1;
            while((resizes[0] > 0)) {
                resizes[0] = 0;
                pairs->copyArrayToHost(19); // nrecv
                change_size_after_exchange0(pairs, particle_capacity, &nlocal, nrecv, resizes);
                const int a492 = resizes[0];
                const bool e1175 = a492 > 0;
                if(e1175) {
                    PAIRS_DEBUG("resizes[0] -> particle_capacity\n");
                    const int a493 = resizes[0];
                    const int e1176 = a493 * 2;
                    particle_capacity = e1176;
                    pairs->reallocArray(9, &particle_cell, &d_particle_cell, (sizeof(int) * particle_capacity));
                    pairs->reallocArray(10, &neighborlists, &d_neighborlists, (sizeof(int) * (particle_capacity * neighborlist_capacity)));
                    pairs->reallocArray(11, &numneighs, &d_numneighs, (sizeof(int) * particle_capacity));
                    pairs->reallocArray(16, &exchg_flag, nullptr, (sizeof(int) * particle_capacity));
                    pairs->reallocProperty(0, &mass, &d_mass, (0 + particle_capacity));
                    pairs->reallocProperty(1, &position, &d_position, (0 + particle_capacity), 3);
                    pairs->reallocProperty(2, &velocity, &d_velocity, (0 + particle_capacity), 3);
                    pairs->reallocProperty(3, &force, &d_force, (0 + particle_capacity), 3);
                }
            }
            nsend_all = 0;
            nghost = 0;
            nsend[0] = 0;
            nrecv[0] = 0;
            send_offsets[0] = 0;
            recv_offsets[0] = 0;
            nsend[1] = 0;
            nrecv[1] = 0;
            send_offsets[1] = 0;
            recv_offsets[1] = 0;
            resizes[0] = 1;
            while((resizes[0] > 0)) {
                resizes[0] = 0;
                pairs->copyArrayToHost(15); // send_map
                pairs->copyArrayToHost(16); // exchg_flag
                pairs->copyArrayToHost(18); // send_mult
                pairs->copyArrayToHost(19); // nrecv
                pairs->copyArrayToHost(12); // nsend
                pairs->copyPropertyToHost(1); // position
                pairs->setArrayHostFlag(15); // send_map
                pairs->clearArrayDeviceFlag(15); // send_map
                pairs->setArrayHostFlag(16); // exchg_flag
                pairs->clearArrayDeviceFlag(16); // exchg_flag
                pairs->setArrayHostFlag(18); // send_mult
                pairs->clearArrayDeviceFlag(18); // send_mult
                pairs->setArrayHostFlag(19); // nrecv
                pairs->clearArrayDeviceFlag(19); // nrecv
                pairs->setArrayHostFlag(12); // nsend
                pairs->clearArrayDeviceFlag(12); // nsend
                determine_exchange_particles1(pairs, nlocal, nghost, send_capacity, &nsend_all, nsend, nrecv, exchg_flag, subdom, pbc, send_map, send_mult, resizes, position);
                const int a497 = resizes[0];
                const bool e1183 = a497 > 0;
                if(e1183) {
                    PAIRS_DEBUG("resizes[0] -> send_capacity\n");
                    const int a498 = resizes[0];
                    const int e1184 = a498 * 2;
                    send_capacity = e1184;
                    pairs->reallocArray(14, &send_buffer, &d_send_buffer, (sizeof(double) * (send_capacity * elem_capacity)));
                    pairs->reallocArray(15, &send_map, &d_send_map, (sizeof(int) * send_capacity));
                    pairs->reallocArray(17, &exchg_copy_to, &d_exchg_copy_to, (sizeof(int) * send_capacity));
                    pairs->reallocArray(18, &send_mult, &d_send_mult, (sizeof(int) * (send_capacity * 3)));
                }
            }
            pairs->communicateSizes(1, nsend, nrecv);
            pairs->copyArrayToHost(19); // nrecv
            pairs->copyArrayToHost(20); // recv_offsets
            pairs->copyArrayToHost(12); // nsend
            pairs->copyArrayToHost(13); // send_offsets
            pairs->setArrayHostFlag(20); // recv_offsets
            pairs->clearArrayDeviceFlag(20); // recv_offsets
            pairs->setArrayHostFlag(13); // send_offsets
            pairs->clearArrayDeviceFlag(13); // send_offsets
            set_communication_offsets1(pairs, nsend, send_offsets, nrecv, recv_offsets);
            pairs->copyArrayToDevice(15); // send_map
            pairs->copyArrayToDevice(13); // send_offsets
            pairs->copyArrayToDevice(18); // send_mult
            pairs->copyArrayToDevice(12); // nsend
            pairs->copyArrayToDevice(14); // send_buffer
            pairs->copyPropertyToDevice(2); // velocity
            pairs->copyPropertyToDevice(0); // mass
            pairs->copyPropertyToDevice(1); // position
            pairs->setArrayDeviceFlag(14); // send_buffer
            pairs->clearArrayHostFlag(14); // send_buffer
            pack_ghost_particles1_0_1_2(pairs, grid0_d0_max, grid0_d0_min, grid0_d1_max, grid0_d1_min, grid0_d2_max, grid0_d2_min, d_send_map, d_send_buffer, d_send_mult, d_send_offsets, send_offsets, d_nsend, nsend, d_mass, d_position, d_velocity);
            pairs->copyArrayToHost(15); // send_map
            pairs->copyArrayToHost(16); // exchg_flag
            pairs->copyArrayToHost(17); // exchg_copy_to
            pairs->setArrayHostFlag(17); // exchg_copy_to
            pairs->clearArrayDeviceFlag(17); // exchg_copy_to
            remove_exchanged_particles_pt1(pairs, nlocal, nsend_all, send_map, exchg_flag, exchg_copy_to);
            pairs->copyArrayToDevice(15); // send_map
            pairs->copyArrayToDevice(17); // exchg_copy_to
            pairs->copyPropertyToDevice(2); // velocity
            pairs->copyPropertyToDevice(0); // mass
            pairs->copyPropertyToDevice(1); // position
            pairs->setPropertyDeviceFlag(2); // velocity
            pairs->clearPropertyHostFlag(2); // velocity
            pairs->setPropertyDeviceFlag(0); // mass
            pairs->clearPropertyHostFlag(0); // mass
            pairs->setPropertyDeviceFlag(1); // position
            pairs->clearPropertyHostFlag(1); // position
            remove_exchanged_particles_pt2(pairs, nsend_all, &nlocal, d_exchg_copy_to, d_send_map, d_mass, d_position, d_velocity);
            pairs->communicateData(1, 7, send_buffer, send_offsets, nsend, recv_buffer, recv_offsets, nrecv);
            pairs->copyArrayToDevice(21); // recv_buffer
            pairs->copyArrayToDevice(19); // nrecv
            pairs->copyArrayToDevice(20); // recv_offsets
            pairs->copyPropertyToDevice(2); // velocity
            pairs->copyPropertyToDevice(0); // mass
            pairs->copyPropertyToDevice(1); // position
            pairs->setPropertyDeviceFlag(2); // velocity
            pairs->clearPropertyHostFlag(2); // velocity
            pairs->setPropertyDeviceFlag(0); // mass
            pairs->clearPropertyHostFlag(0); // mass
            pairs->setPropertyDeviceFlag(1); // position
            pairs->clearPropertyHostFlag(1); // position
            unpack_ghost_particles1_0_1_2(pairs, nlocal, d_recv_buffer, d_recv_offsets, recv_offsets, d_nrecv, nrecv, d_mass, d_position, d_velocity);
            resizes[0] = 1;
            while((resizes[0] > 0)) {
                resizes[0] = 0;
                pairs->copyArrayToHost(19); // nrecv
                change_size_after_exchange1(pairs, particle_capacity, &nlocal, nrecv, resizes);
                const int a502 = resizes[0];
                const bool e1196 = a502 > 0;
                if(e1196) {
                    PAIRS_DEBUG("resizes[0] -> particle_capacity\n");
                    const int a503 = resizes[0];
                    const int e1197 = a503 * 2;
                    particle_capacity = e1197;
                    pairs->reallocArray(9, &particle_cell, &d_particle_cell, (sizeof(int) * particle_capacity));
                    pairs->reallocArray(10, &neighborlists, &d_neighborlists, (sizeof(int) * (particle_capacity * neighborlist_capacity)));
                    pairs->reallocArray(11, &numneighs, &d_numneighs, (sizeof(int) * particle_capacity));
                    pairs->reallocArray(16, &exchg_flag, nullptr, (sizeof(int) * particle_capacity));
                    pairs->reallocProperty(0, &mass, &d_mass, (0 + particle_capacity));
                    pairs->reallocProperty(1, &position, &d_position, (0 + particle_capacity), 3);
                    pairs->reallocProperty(2, &velocity, &d_velocity, (0 + particle_capacity), 3);
                    pairs->reallocProperty(3, &force, &d_force, (0 + particle_capacity), 3);
                }
            }
            nsend_all = 0;
            nghost = 0;
            nsend[0] = 0;
            nrecv[0] = 0;
            send_offsets[0] = 0;
            recv_offsets[0] = 0;
            nsend[1] = 0;
            nrecv[1] = 0;
            send_offsets[1] = 0;
            recv_offsets[1] = 0;
            nsend[2] = 0;
            nrecv[2] = 0;
            send_offsets[2] = 0;
            recv_offsets[2] = 0;
            nsend[3] = 0;
            nrecv[3] = 0;
            send_offsets[3] = 0;
            recv_offsets[3] = 0;
            resizes[0] = 1;
            while((resizes[0] > 0)) {
                resizes[0] = 0;
                pairs->copyArrayToHost(15); // send_map
                pairs->copyArrayToHost(16); // exchg_flag
                pairs->copyArrayToHost(18); // send_mult
                pairs->copyArrayToHost(19); // nrecv
                pairs->copyArrayToHost(12); // nsend
                pairs->copyPropertyToHost(1); // position
                pairs->setArrayHostFlag(15); // send_map
                pairs->clearArrayDeviceFlag(15); // send_map
                pairs->setArrayHostFlag(16); // exchg_flag
                pairs->clearArrayDeviceFlag(16); // exchg_flag
                pairs->setArrayHostFlag(18); // send_mult
                pairs->clearArrayDeviceFlag(18); // send_mult
                pairs->setArrayHostFlag(19); // nrecv
                pairs->clearArrayDeviceFlag(19); // nrecv
                pairs->setArrayHostFlag(12); // nsend
                pairs->clearArrayDeviceFlag(12); // nsend
                determine_exchange_particles2(pairs, nlocal, nghost, send_capacity, &nsend_all, nsend, nrecv, exchg_flag, subdom, pbc, send_map, send_mult, resizes, position);
                const int a507 = resizes[0];
                const bool e1204 = a507 > 0;
                if(e1204) {
                    PAIRS_DEBUG("resizes[0] -> send_capacity\n");
                    const int a508 = resizes[0];
                    const int e1205 = a508 * 2;
                    send_capacity = e1205;
                    pairs->reallocArray(14, &send_buffer, &d_send_buffer, (sizeof(double) * (send_capacity * elem_capacity)));
                    pairs->reallocArray(15, &send_map, &d_send_map, (sizeof(int) * send_capacity));
                    pairs->reallocArray(17, &exchg_copy_to, &d_exchg_copy_to, (sizeof(int) * send_capacity));
                    pairs->reallocArray(18, &send_mult, &d_send_mult, (sizeof(int) * (send_capacity * 3)));
                }
            }
            pairs->communicateSizes(2, nsend, nrecv);
            pairs->copyArrayToHost(19); // nrecv
            pairs->copyArrayToHost(20); // recv_offsets
            pairs->copyArrayToHost(12); // nsend
            pairs->copyArrayToHost(13); // send_offsets
            pairs->setArrayHostFlag(20); // recv_offsets
            pairs->clearArrayDeviceFlag(20); // recv_offsets
            pairs->setArrayHostFlag(13); // send_offsets
            pairs->clearArrayDeviceFlag(13); // send_offsets
            set_communication_offsets2(pairs, nsend, send_offsets, nrecv, recv_offsets);
            pairs->copyArrayToDevice(15); // send_map
            pairs->copyArrayToDevice(13); // send_offsets
            pairs->copyArrayToDevice(18); // send_mult
            pairs->copyArrayToDevice(12); // nsend
            pairs->copyArrayToDevice(14); // send_buffer
            pairs->copyPropertyToDevice(2); // velocity
            pairs->copyPropertyToDevice(0); // mass
            pairs->copyPropertyToDevice(1); // position
            pairs->setArrayDeviceFlag(14); // send_buffer
            pairs->clearArrayHostFlag(14); // send_buffer
            pack_ghost_particles2_0_1_2(pairs, grid0_d0_max, grid0_d0_min, grid0_d1_max, grid0_d1_min, grid0_d2_max, grid0_d2_min, d_send_map, d_send_buffer, d_send_mult, d_send_offsets, send_offsets, d_nsend, nsend, d_mass, d_position, d_velocity);
            pairs->copyArrayToHost(15); // send_map
            pairs->copyArrayToHost(16); // exchg_flag
            pairs->copyArrayToHost(17); // exchg_copy_to
            pairs->setArrayHostFlag(17); // exchg_copy_to
            pairs->clearArrayDeviceFlag(17); // exchg_copy_to
            remove_exchanged_particles_pt1(pairs, nlocal, nsend_all, send_map, exchg_flag, exchg_copy_to);
            pairs->copyArrayToDevice(15); // send_map
            pairs->copyArrayToDevice(17); // exchg_copy_to
            pairs->copyPropertyToDevice(2); // velocity
            pairs->copyPropertyToDevice(0); // mass
            pairs->copyPropertyToDevice(1); // position
            pairs->setPropertyDeviceFlag(2); // velocity
            pairs->clearPropertyHostFlag(2); // velocity
            pairs->setPropertyDeviceFlag(0); // mass
            pairs->clearPropertyHostFlag(0); // mass
            pairs->setPropertyDeviceFlag(1); // position
            pairs->clearPropertyHostFlag(1); // position
            remove_exchanged_particles_pt2(pairs, nsend_all, &nlocal, d_exchg_copy_to, d_send_map, d_mass, d_position, d_velocity);
            pairs->communicateData(2, 7, send_buffer, send_offsets, nsend, recv_buffer, recv_offsets, nrecv);
            pairs->copyArrayToDevice(21); // recv_buffer
            pairs->copyArrayToDevice(19); // nrecv
            pairs->copyArrayToDevice(20); // recv_offsets
            pairs->copyPropertyToDevice(2); // velocity
            pairs->copyPropertyToDevice(0); // mass
            pairs->copyPropertyToDevice(1); // position
            pairs->setPropertyDeviceFlag(2); // velocity
            pairs->clearPropertyHostFlag(2); // velocity
            pairs->setPropertyDeviceFlag(0); // mass
            pairs->clearPropertyHostFlag(0); // mass
            pairs->setPropertyDeviceFlag(1); // position
            pairs->clearPropertyHostFlag(1); // position
            unpack_ghost_particles2_0_1_2(pairs, nlocal, d_recv_buffer, d_recv_offsets, recv_offsets, d_nrecv, nrecv, d_mass, d_position, d_velocity);
            resizes[0] = 1;
            while((resizes[0] > 0)) {
                resizes[0] = 0;
                pairs->copyArrayToHost(19); // nrecv
                change_size_after_exchange2(pairs, particle_capacity, &nlocal, nrecv, resizes);
                const int a512 = resizes[0];
                const bool e1217 = a512 > 0;
                if(e1217) {
                    PAIRS_DEBUG("resizes[0] -> particle_capacity\n");
                    const int a513 = resizes[0];
                    const int e1218 = a513 * 2;
                    particle_capacity = e1218;
                    pairs->reallocArray(9, &particle_cell, &d_particle_cell, (sizeof(int) * particle_capacity));
                    pairs->reallocArray(10, &neighborlists, &d_neighborlists, (sizeof(int) * (particle_capacity * neighborlist_capacity)));
                    pairs->reallocArray(11, &numneighs, &d_numneighs, (sizeof(int) * particle_capacity));
                    pairs->reallocArray(16, &exchg_flag, nullptr, (sizeof(int) * particle_capacity));
                    pairs->reallocProperty(0, &mass, &d_mass, (0 + particle_capacity));
                    pairs->reallocProperty(1, &position, &d_position, (0 + particle_capacity), 3);
                    pairs->reallocProperty(2, &velocity, &d_velocity, (0 + particle_capacity), 3);
                    pairs->reallocProperty(3, &force, &d_force, (0 + particle_capacity), 3);
                }
            }
        }
        if(((i1 % 20) == 0)) {
            nsend_all = 0;
            nghost = 0;
            resizes[0] = 1;
            while((resizes[0] > 0)) {
                resizes[0] = 0;
                pairs->copyArrayToHost(15); // send_map
                pairs->copyArrayToHost(19); // nrecv
                pairs->copyArrayToHost(12); // nsend
                pairs->copyArrayToHost(18); // send_mult
                pairs->copyPropertyToHost(1); // position
                pairs->setArrayHostFlag(15); // send_map
                pairs->clearArrayDeviceFlag(15); // send_map
                pairs->setArrayHostFlag(19); // nrecv
                pairs->clearArrayDeviceFlag(19); // nrecv
                pairs->setArrayHostFlag(12); // nsend
                pairs->clearArrayDeviceFlag(12); // nsend
                pairs->setArrayHostFlag(18); // send_mult
                pairs->clearArrayDeviceFlag(18); // send_mult
                determine_ghost_particles0(pairs, nlocal, nghost, send_capacity, &nsend_all, nsend, nrecv, subdom, pbc, send_map, send_mult, resizes, position);
                const int a517 = resizes[0];
                const bool e1225 = a517 > 0;
                if(e1225) {
                    PAIRS_DEBUG("resizes[0] -> send_capacity\n");
                    const int a518 = resizes[0];
                    const int e1226 = a518 * 2;
                    send_capacity = e1226;
                    pairs->reallocArray(14, &send_buffer, &d_send_buffer, (sizeof(double) * (send_capacity * elem_capacity)));
                    pairs->reallocArray(15, &send_map, &d_send_map, (sizeof(int) * send_capacity));
                    pairs->reallocArray(17, &exchg_copy_to, &d_exchg_copy_to, (sizeof(int) * send_capacity));
                    pairs->reallocArray(18, &send_mult, &d_send_mult, (sizeof(int) * (send_capacity * 3)));
                }
            }
            pairs->communicateSizes(0, nsend, nrecv);
            pairs->copyArrayToHost(19); // nrecv
            pairs->copyArrayToHost(20); // recv_offsets
            pairs->copyArrayToHost(12); // nsend
            pairs->copyArrayToHost(13); // send_offsets
            pairs->setArrayHostFlag(20); // recv_offsets
            pairs->clearArrayDeviceFlag(20); // recv_offsets
            pairs->setArrayHostFlag(13); // send_offsets
            pairs->clearArrayDeviceFlag(13); // send_offsets
            set_communication_offsets0(pairs, send_offsets, recv_offsets, nsend, nrecv);
            pairs->copyArrayToDevice(15); // send_map
            pairs->copyArrayToDevice(13); // send_offsets
            pairs->copyArrayToDevice(18); // send_mult
            pairs->copyArrayToDevice(12); // nsend
            pairs->copyArrayToDevice(14); // send_buffer
            pairs->copyPropertyToDevice(0); // mass
            pairs->copyPropertyToDevice(1); // position
            pairs->setArrayDeviceFlag(14); // send_buffer
            pairs->clearArrayHostFlag(14); // send_buffer
            pack_ghost_particles0_0_1(pairs, grid0_d0_max, grid0_d0_min, grid0_d1_max, grid0_d1_min, grid0_d2_max, grid0_d2_min, d_send_map, d_send_buffer, d_send_mult, d_send_offsets, send_offsets, d_nsend, nsend, d_mass, d_position);
            pairs->communicateData(0, 4, send_buffer, send_offsets, nsend, recv_buffer, recv_offsets, nrecv);
            pairs->copyArrayToDevice(21); // recv_buffer
            pairs->copyArrayToDevice(19); // nrecv
            pairs->copyArrayToDevice(20); // recv_offsets
            pairs->copyPropertyToDevice(0); // mass
            pairs->copyPropertyToDevice(1); // position
            pairs->setPropertyDeviceFlag(0); // mass
            pairs->clearPropertyHostFlag(0); // mass
            pairs->setPropertyDeviceFlag(1); // position
            pairs->clearPropertyHostFlag(1); // position
            unpack_ghost_particles0_0_1(pairs, nlocal, d_recv_buffer, d_recv_offsets, recv_offsets, d_nrecv, nrecv, d_mass, d_position);
            const int a24 = nrecv[0];
            const int a25 = nrecv[1];
            const int e66 = a24 + a25;
            const int e67 = nghost + e66;
            nghost = e67;
            resizes[0] = 1;
            while((resizes[0] > 0)) {
                resizes[0] = 0;
                pairs->copyArrayToHost(15); // send_map
                pairs->copyArrayToHost(19); // nrecv
                pairs->copyArrayToHost(12); // nsend
                pairs->copyArrayToHost(18); // send_mult
                pairs->copyPropertyToHost(1); // position
                pairs->setArrayHostFlag(15); // send_map
                pairs->clearArrayDeviceFlag(15); // send_map
                pairs->setArrayHostFlag(19); // nrecv
                pairs->clearArrayDeviceFlag(19); // nrecv
                pairs->setArrayHostFlag(12); // nsend
                pairs->clearArrayDeviceFlag(12); // nsend
                pairs->setArrayHostFlag(18); // send_mult
                pairs->clearArrayDeviceFlag(18); // send_mult
                determine_ghost_particles1(pairs, nlocal, nghost, send_capacity, &nsend_all, nsend, nrecv, subdom, pbc, send_map, send_mult, resizes, position);
                const int a522 = resizes[0];
                const bool e1234 = a522 > 0;
                if(e1234) {
                    PAIRS_DEBUG("resizes[0] -> send_capacity\n");
                    const int a523 = resizes[0];
                    const int e1235 = a523 * 2;
                    send_capacity = e1235;
                    pairs->reallocArray(14, &send_buffer, &d_send_buffer, (sizeof(double) * (send_capacity * elem_capacity)));
                    pairs->reallocArray(15, &send_map, &d_send_map, (sizeof(int) * send_capacity));
                    pairs->reallocArray(17, &exchg_copy_to, &d_exchg_copy_to, (sizeof(int) * send_capacity));
                    pairs->reallocArray(18, &send_mult, &d_send_mult, (sizeof(int) * (send_capacity * 3)));
                }
            }
            pairs->communicateSizes(1, nsend, nrecv);
            pairs->copyArrayToHost(19); // nrecv
            pairs->copyArrayToHost(20); // recv_offsets
            pairs->copyArrayToHost(12); // nsend
            pairs->copyArrayToHost(13); // send_offsets
            pairs->setArrayHostFlag(20); // recv_offsets
            pairs->clearArrayDeviceFlag(20); // recv_offsets
            pairs->setArrayHostFlag(13); // send_offsets
            pairs->clearArrayDeviceFlag(13); // send_offsets
            set_communication_offsets1(pairs, nsend, send_offsets, nrecv, recv_offsets);
            pairs->copyArrayToDevice(15); // send_map
            pairs->copyArrayToDevice(13); // send_offsets
            pairs->copyArrayToDevice(18); // send_mult
            pairs->copyArrayToDevice(12); // nsend
            pairs->copyArrayToDevice(14); // send_buffer
            pairs->copyPropertyToDevice(0); // mass
            pairs->copyPropertyToDevice(1); // position
            pairs->setArrayDeviceFlag(14); // send_buffer
            pairs->clearArrayHostFlag(14); // send_buffer
            pack_ghost_particles1_0_1(pairs, grid0_d0_max, grid0_d0_min, grid0_d1_max, grid0_d1_min, grid0_d2_max, grid0_d2_min, d_send_map, d_send_buffer, d_send_mult, d_send_offsets, send_offsets, d_nsend, nsend, d_mass, d_position);
            pairs->communicateData(1, 4, send_buffer, send_offsets, nsend, recv_buffer, recv_offsets, nrecv);
            pairs->copyArrayToDevice(21); // recv_buffer
            pairs->copyArrayToDevice(19); // nrecv
            pairs->copyArrayToDevice(20); // recv_offsets
            pairs->copyPropertyToDevice(0); // mass
            pairs->copyPropertyToDevice(1); // position
            pairs->setPropertyDeviceFlag(0); // mass
            pairs->clearPropertyHostFlag(0); // mass
            pairs->setPropertyDeviceFlag(1); // position
            pairs->clearPropertyHostFlag(1); // position
            unpack_ghost_particles1_0_1(pairs, nlocal, d_recv_buffer, d_recv_offsets, recv_offsets, d_nrecv, nrecv, d_mass, d_position);
            const int a26 = nrecv[2];
            const int a27 = nrecv[3];
            const int e69 = a26 + a27;
            const int e70 = nghost + e69;
            nghost = e70;
            resizes[0] = 1;
            while((resizes[0] > 0)) {
                resizes[0] = 0;
                pairs->copyArrayToHost(15); // send_map
                pairs->copyArrayToHost(19); // nrecv
                pairs->copyArrayToHost(12); // nsend
                pairs->copyArrayToHost(18); // send_mult
                pairs->copyPropertyToHost(1); // position
                pairs->setArrayHostFlag(15); // send_map
                pairs->clearArrayDeviceFlag(15); // send_map
                pairs->setArrayHostFlag(19); // nrecv
                pairs->clearArrayDeviceFlag(19); // nrecv
                pairs->setArrayHostFlag(12); // nsend
                pairs->clearArrayDeviceFlag(12); // nsend
                pairs->setArrayHostFlag(18); // send_mult
                pairs->clearArrayDeviceFlag(18); // send_mult
                determine_ghost_particles2(pairs, nlocal, nghost, send_capacity, &nsend_all, nsend, nrecv, subdom, pbc, send_map, send_mult, resizes, position);
                const int a527 = resizes[0];
                const bool e1243 = a527 > 0;
                if(e1243) {
                    PAIRS_DEBUG("resizes[0] -> send_capacity\n");
                    const int a528 = resizes[0];
                    const int e1244 = a528 * 2;
                    send_capacity = e1244;
                    pairs->reallocArray(14, &send_buffer, &d_send_buffer, (sizeof(double) * (send_capacity * elem_capacity)));
                    pairs->reallocArray(15, &send_map, &d_send_map, (sizeof(int) * send_capacity));
                    pairs->reallocArray(17, &exchg_copy_to, &d_exchg_copy_to, (sizeof(int) * send_capacity));
                    pairs->reallocArray(18, &send_mult, &d_send_mult, (sizeof(int) * (send_capacity * 3)));
                }
            }
            pairs->communicateSizes(2, nsend, nrecv);
            pairs->copyArrayToHost(19); // nrecv
            pairs->copyArrayToHost(20); // recv_offsets
            pairs->copyArrayToHost(12); // nsend
            pairs->copyArrayToHost(13); // send_offsets
            pairs->setArrayHostFlag(20); // recv_offsets
            pairs->clearArrayDeviceFlag(20); // recv_offsets
            pairs->setArrayHostFlag(13); // send_offsets
            pairs->clearArrayDeviceFlag(13); // send_offsets
            set_communication_offsets2(pairs, nsend, send_offsets, nrecv, recv_offsets);
            pairs->copyArrayToDevice(15); // send_map
            pairs->copyArrayToDevice(13); // send_offsets
            pairs->copyArrayToDevice(18); // send_mult
            pairs->copyArrayToDevice(12); // nsend
            pairs->copyArrayToDevice(14); // send_buffer
            pairs->copyPropertyToDevice(0); // mass
            pairs->copyPropertyToDevice(1); // position
            pairs->setArrayDeviceFlag(14); // send_buffer
            pairs->clearArrayHostFlag(14); // send_buffer
            pack_ghost_particles2_0_1(pairs, grid0_d0_max, grid0_d0_min, grid0_d1_max, grid0_d1_min, grid0_d2_max, grid0_d2_min, d_send_map, d_send_buffer, d_send_mult, d_send_offsets, send_offsets, d_nsend, nsend, d_mass, d_position);
            pairs->communicateData(2, 4, send_buffer, send_offsets, nsend, recv_buffer, recv_offsets, nrecv);
            pairs->copyArrayToDevice(21); // recv_buffer
            pairs->copyArrayToDevice(19); // nrecv
            pairs->copyArrayToDevice(20); // recv_offsets
            pairs->copyPropertyToDevice(0); // mass
            pairs->copyPropertyToDevice(1); // position
            pairs->setPropertyDeviceFlag(0); // mass
            pairs->clearPropertyHostFlag(0); // mass
            pairs->setPropertyDeviceFlag(1); // position
            pairs->clearPropertyHostFlag(1); // position
            unpack_ghost_particles2_0_1(pairs, nlocal, d_recv_buffer, d_recv_offsets, recv_offsets, d_nrecv, nrecv, d_mass, d_position);
            const int a28 = nrecv[4];
            const int a29 = nrecv[5];
            const int e72 = a28 + a29;
            const int e73 = nghost + e72;
            nghost = e73;
        } else {
            pairs->copyArrayToDevice(15); // send_map
            pairs->copyArrayToDevice(13); // send_offsets
            pairs->copyArrayToDevice(18); // send_mult
            pairs->copyArrayToDevice(12); // nsend
            pairs->copyArrayToDevice(14); // send_buffer
            pairs->copyPropertyToDevice(1); // position
            pairs->setArrayDeviceFlag(14); // send_buffer
            pairs->clearArrayHostFlag(14); // send_buffer
            pack_ghost_particles0_1(pairs, grid0_d0_max, grid0_d0_min, grid0_d1_max, grid0_d1_min, grid0_d2_max, grid0_d2_min, d_send_map, d_send_mult, d_send_buffer, d_send_offsets, send_offsets, d_nsend, nsend, d_position);
            pairs->communicateData(0, 3, send_buffer, send_offsets, nsend, recv_buffer, recv_offsets, nrecv);
            pairs->copyArrayToDevice(21); // recv_buffer
            pairs->copyArrayToDevice(19); // nrecv
            pairs->copyArrayToDevice(20); // recv_offsets
            pairs->copyPropertyToDevice(1); // position
            pairs->setPropertyDeviceFlag(1); // position
            pairs->clearPropertyHostFlag(1); // position
            unpack_ghost_particles0_1(pairs, nlocal, d_recv_buffer, d_recv_offsets, recv_offsets, d_nrecv, nrecv, d_position);
            pairs->copyArrayToDevice(15); // send_map
            pairs->copyArrayToDevice(13); // send_offsets
            pairs->copyArrayToDevice(18); // send_mult
            pairs->copyArrayToDevice(12); // nsend
            pairs->copyArrayToDevice(14); // send_buffer
            pairs->copyPropertyToDevice(1); // position
            pairs->setArrayDeviceFlag(14); // send_buffer
            pairs->clearArrayHostFlag(14); // send_buffer
            pack_ghost_particles1_1(pairs, grid0_d0_max, grid0_d0_min, grid0_d1_max, grid0_d1_min, grid0_d2_max, grid0_d2_min, d_send_map, d_send_mult, d_send_buffer, d_send_offsets, send_offsets, d_nsend, nsend, d_position);
            pairs->communicateData(1, 3, send_buffer, send_offsets, nsend, recv_buffer, recv_offsets, nrecv);
            pairs->copyArrayToDevice(21); // recv_buffer
            pairs->copyArrayToDevice(19); // nrecv
            pairs->copyArrayToDevice(20); // recv_offsets
            pairs->copyPropertyToDevice(1); // position
            pairs->setPropertyDeviceFlag(1); // position
            pairs->clearPropertyHostFlag(1); // position
            unpack_ghost_particles1_1(pairs, nlocal, d_recv_buffer, d_recv_offsets, recv_offsets, d_nrecv, nrecv, d_position);
            pairs->copyArrayToDevice(15); // send_map
            pairs->copyArrayToDevice(13); // send_offsets
            pairs->copyArrayToDevice(18); // send_mult
            pairs->copyArrayToDevice(12); // nsend
            pairs->copyArrayToDevice(14); // send_buffer
            pairs->copyPropertyToDevice(1); // position
            pairs->setArrayDeviceFlag(14); // send_buffer
            pairs->clearArrayHostFlag(14); // send_buffer
            pack_ghost_particles2_1(pairs, grid0_d0_max, grid0_d0_min, grid0_d1_max, grid0_d1_min, grid0_d2_max, grid0_d2_min, d_send_map, d_send_mult, d_send_buffer, d_send_offsets, send_offsets, d_nsend, nsend, d_position);
            pairs->communicateData(2, 3, send_buffer, send_offsets, nsend, recv_buffer, recv_offsets, nrecv);
            pairs->copyArrayToDevice(21); // recv_buffer
            pairs->copyArrayToDevice(19); // nrecv
            pairs->copyArrayToDevice(20); // recv_offsets
            pairs->copyPropertyToDevice(1); // position
            pairs->setPropertyDeviceFlag(1); // position
            pairs->clearPropertyHostFlag(1); // position
            unpack_ghost_particles2_1(pairs, nlocal, d_recv_buffer, d_recv_offsets, recv_offsets, d_nrecv, nrecv, d_position);
        }
        if(((i1 % 20) == 0)) {
            resizes[0] = 1;
            while((resizes[0] > 0)) {
                resizes[0] = 0;
                pairs->copyArrayToDevice(7); // cell_sizes
                pairs->copyArrayToDevice(9); // particle_cell
                pairs->copyArrayToDevice(6); // cell_particles
                pairs->copyPropertyToDevice(1); // position
                pairs->setArrayDeviceFlag(7); // cell_sizes
                pairs->clearArrayHostFlag(7); // cell_sizes
                pairs->setArrayDeviceFlag(9); // particle_cell
                pairs->clearArrayHostFlag(9); // particle_cell
                pairs->setArrayDeviceFlag(6); // cell_particles
                pairs->clearArrayHostFlag(6); // cell_particles
                pairs->copyArrayToDevice(0); // resizes
                build_cell_lists(pairs, ncells, nlocal, nghost, grid0_d0_min, grid0_d1_min, grid0_d2_min, cell_capacity, d_cell_sizes, d_dim_cells, d_particle_cell, d_resizes, d_cell_particles, d_position);
                pairs->copyArrayToHost(0); // resizes
                const int a532 = resizes[0];
                const bool e1252 = a532 > 0;
                if(e1252) {
                    PAIRS_DEBUG("resizes[0] -> cell_capacity\n");
                    const int a533 = resizes[0];
                    const int e1253 = a533 * 2;
                    cell_capacity = e1253;
                    pairs->reallocArray(6, &cell_particles, &d_cell_particles, (sizeof(int) * (ncells_capacity * cell_capacity)));
                }
            }
        }
        if(((i1 % 20) == 0)) {
            resizes[0] = 1;
            while((resizes[0] > 0)) {
                resizes[0] = 0;
                pairs->copyArrayToDevice(10); // neighborlists
                pairs->copyArrayToDevice(11); // numneighs
                pairs->copyArrayToDevice(6); // cell_particles
                pairs->copyArrayToDevice(7); // cell_sizes
                pairs->copyArrayToDevice(8); // stencil
                pairs->copyArrayToDevice(9); // particle_cell
                pairs->copyPropertyToDevice(1); // position
                pairs->setArrayDeviceFlag(10); // neighborlists
                pairs->clearArrayHostFlag(10); // neighborlists
                pairs->setArrayDeviceFlag(11); // numneighs
                pairs->clearArrayHostFlag(11); // numneighs
                pairs->copyArrayToDevice(0); // resizes
                neighbor_lists_build(pairs, nlocal, ncells, cell_capacity, neighborlist_capacity, nstencil, d_numneighs, d_particle_cell, d_stencil, d_cell_sizes, d_cell_particles, d_neighborlists, d_resizes, d_position);
                pairs->copyArrayToHost(0); // resizes
                const int a537 = resizes[0];
                const bool e1257 = a537 > 0;
                if(e1257) {
                    PAIRS_DEBUG("resizes[0] -> neighborlist_capacity\n");
                    const int a538 = resizes[0];
                    const int e1258 = a538 * 2;
                    neighborlist_capacity = e1258;
                    pairs->reallocArray(10, &neighborlists, &d_neighborlists, (sizeof(int) * (particle_capacity * neighborlist_capacity)));
                }
            }
        }
        pairs->copyPropertyToDevice(3); // force
        pairs->setPropertyDeviceFlag(3); // force
        pairs->clearPropertyHostFlag(3); // force
        reset_volatile_properties(pairs, nlocal, d_force);
        pairs->copyArrayToDevice(10); // neighborlists
        pairs->copyArrayToDevice(11); // numneighs
        pairs->copyPropertyToDevice(3); // force
        pairs->copyPropertyToDevice(1); // position
        pairs->setPropertyDeviceFlag(3); // force
        pairs->clearPropertyHostFlag(3); // force
        lj(pairs, neighborlist_capacity, nlocal, d_numneighs, d_neighborlists, d_position, d_force);
        pairs->copyPropertyToDevice(2); // velocity
        pairs->copyPropertyToDevice(0); // mass
        pairs->copyPropertyToDevice(3); // force
        pairs->copyPropertyToDevice(1); // position
        pairs->setPropertyDeviceFlag(2); // velocity
        pairs->clearPropertyHostFlag(2); // velocity
        pairs->setPropertyDeviceFlag(1); // position
        pairs->clearPropertyHostFlag(1); // position
        euler(pairs, nlocal, d_velocity, d_force, d_mass, d_position);
        const int e82 = i1 + 1;
        pairs::vtk_write_data(pairs, "output/test_gpu_local", 0, nlocal, e82);
        const int e1005 = nlocal + nghost;
        pairs::vtk_write_data(pairs, "output/test_gpu_ghost", nlocal, e1005, e82);
    }
    delete pairs;
    return 0;
}
