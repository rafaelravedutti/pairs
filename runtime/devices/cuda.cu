#include <hip/hip_runtime.h>
#include <iostream>

#define CUDA_ASSERT(a) { pairs::cuda_assert((a), __FILE__, __LINE__); }

namespace pairs {

inline void cuda_assert(hipError_t err, const char *file, int line) {
    if(err != hipSuccess) {
        std::cerr << file << ":" << line << ": " << hipGetErrorString(err) << std::endl;
        exit(-1);
    }
}

__host__ void *device_alloc(size_t size) {
    void *ptr;
    CUDA_ASSERT(hipMalloc(&ptr, size));
    return ptr;
}

__host__ void *device_realloc(void *ptr, size_t size) {
    void *new_ptr;
    CUDA_ASSERT(hipFree(ptr));
    CUDA_ASSERT(hipMalloc(&new_ptr, size));
    return new_ptr;
}

__host__ void device_free(void *ptr) {
    CUDA_ASSERT(hipFree(ptr));
}

__host__ void copy_to_device(const void *h_ptr, void *d_ptr, size_t count) {
    CUDA_ASSERT(hipMemcpy(d_ptr, h_ptr, count, hipMemcpyHostToDevice));
}

__host__ void copy_to_host(const void *d_ptr, void *h_ptr, size_t count) {
    CUDA_ASSERT(hipMemcpy(h_ptr, d_ptr, count, hipMemcpyDeviceToHost));
}

__host__ void copy_static_symbol_to_device(void *h_ptr, const void *d_ptr, size_t count) {
    CUDA_ASSERT(hipMemcpyToSymbol(HIP_SYMBOL(d_ptr), h_ptr, count));
}

__host__ void copy_static_symbol_to_host(void *d_ptr, const void *h_ptr, size_t count) {
    //CUDA_ASSERT(cudaMemcpyFromSymbol(h_ptr, d_ptr, count));
}

}
